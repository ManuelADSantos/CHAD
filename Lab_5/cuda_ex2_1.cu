﻿// nvcc -o cuda_ex2_1 cuda_ex2_1.cu
//

#include <hip/hip_runtime.h>
#include <stdio.h>  
#include <stdlib.h> 
#include <math.h> 
#include <time.h> 

#define N 1234

__global__ void somaCuda(int* v1, int *result)
{
    int tid = threadIdx.x;
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    result[tid] = v1[index];
        
    for (int s = 1; s < blockDim.x; s *= 2) {
	if (tid % (2*s) == 0) {
	    result[tid] += result[tid + s];
	}
	__syncthreads();
    }
}

int main( int argc, char** argv){

    //Criar Imagem com valores random
    //int N = 2048;
    int result = 0;
    int result2 = 0;
    int *v1 = (int*)malloc(N * sizeof(int));
    int *vOut = (int*)malloc(N * sizeof(int));

    for (int i = 0; i < N; i++){
	v1[i] = i;
	vOut[i] = 0;
    }


    // Versão sequencial
    for (int i = 0; i < N; i++){
	result += v1[i];
    }
	
    printf("Resultado da Soma - Sequencial: %d \n", result);
    
    
    // Versão paralela
    
    int threadsPerBlock = 256; 
    int blocksPerGrid = N/256;

    int *v1Cuda = NULL;
    int *vOutCuda = NULL;

    hipMalloc(&v1Cuda, N * sizeof(int));
    hipMalloc(&vOutCuda, N * sizeof(int));

    hipMemcpy(v1Cuda, v1, N * sizeof(int), hipMemcpyHostToDevice);
    
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);

    somaCuda<<<blocksPerGrid, threadsPerBlock>>>(v1Cuda, vOutCuda);
    
    clock_gettime(CLOCK_MONOTONIC, &end);
    
    hipMemcpy(vOut, vOutCuda, N * sizeof(int), hipMemcpyDeviceToHost);
    
    for(int i = 0; i < blocksPerGrid; i++){
    	result2 += vOut[i];
    }

    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
    double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    
    
    printf("Exercício 2.1 - CUDA sem otimizações: %f ms\n", (finalTime - initialTime));
    printf("Resultado da Soma - CUDA sem otimizações: %d \n", result);

    hipFree(v1Cuda);
    hipFree(vOutCuda);
    
    free(v1);
    free(vOut);

    return 0;
}

