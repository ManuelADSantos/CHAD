#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex3_1_GPU ex3_1_GPU.cu
// -> Run
// ./ex3_1_GPU width height
// ============================================================================

#include <stdio.h>

// ===================== Kernel =====================
__global__ void reduce(int *inData, int *outData, int width, int height)
{   
    // ===== Calculate index 
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    
    // ===== Transpose
    outData[col * height + row] = inData[row * width + col];
}


// ===================== Main =====================
int main(int argc, char *argv[])
{   
    // ===== Check arguments
    if (argc != 3)
	{
		printf("./ex3_1_GPU width height \n");
		return -1;
	}

	// ===== Get width and height
	int width, height;
	width = atoi(argv[1]);
	height = atoi(argv[2]);

    // ===== Declare timer variables
	struct timespec start, end;

	// ===== Get total number of elements
	int N = width * height;

    // ===== Show number of elements
	printf("\n-> Dimensions: 		%d x %d\n", width, height);
	printf("-> Number of elements:  %d\n\n", N);

    // ===== Initialize host matrix
    int host_original[N];
    for(int i=0; i<height; i++)
	{
		for (int j=0; j<width; j++)
		{
			host_original[i*width + j] = i;
		}
	} 
    
    // ===== Allocate device memory
    int *device_original = NULL, *device_transposed = NULL, err = 0;

    err = hipMalloc(&device_original, sizeof(int)*N);
    err = hipMalloc(&device_transposed, sizeof(int)*N);
    if(err != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }
    
	// ===== Get initial time
	clock_gettime(CLOCK_MONOTONIC, &start);

    // ===== Copy data from host memory to device memory
    err = hipMemcpy(device_original,&host_original,sizeof(int)*N,hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }

    // ===== Define block and grid dimensions
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid(width/16, height/16, 1);
    
    // ===== Launch kernel
    reduce<<<blocksPerGrid,threadsPerBlock>>>(device_original, device_transposed, width, height);
    
    // ===== Copy data from device memory to host memory
    int * host_transposed = (int *)malloc(sizeof(int)*N);
    err = hipMemcpy(host_transposed,device_transposed,sizeof(int)*N,hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
	
    // ===== Calculate the elapsed time
    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);~

    // ===== Validate results
    printf("host_transposed[%i][%i]=%u\n", width/2, height/4, host_transposed[((width/2) * height) + height/4]);
    printf("host_transposed[%i][%i]=%u\n", width/4, 3*height/4, host_transposed[((width/4) * height) + 3*height/4]);
    printf("host_transposed[%i][%i]=%u\n", 3*width/4, height/2, host_transposed[((3*width/4) * height) + height/2]);
    
    // ===== Print time
	printf("\n--> Execution Time:\t%f ms\n\n", (finalTime - initialTime));

    // ===== Free device buffers
    hipFree(device_original);
    hipFree(device_transposed);

    // ===== Free host buffers
    free(host_transposed);

    return 0;
}
