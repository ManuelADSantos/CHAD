#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex3_2_GPU ex3_2_GPU.cu
// -> Run
// ./ex3_2_GPU width height
// ============================================================================

#include <stdio.h>

#define TILE_DIM 32
#define BLOCK_ROWS 8

// ===================== Kernel =====================
__global__ void reduce(int *inData, int *outData)
{
    // ===== Shared memory
    __shared__ int tile[TILE_DIM][TILE_DIM];

    // ===== Global memory index
    int x = threadIdx.x + blockIdx.x * TILE_DIM;
    int y = threadIdx.y + blockIdx.y * TILE_DIM;
    int width = gridDim.x * TILE_DIM;

    // ===== Coalesced access to global memory
    for (int j=0; j<TILE_DIM; j+=BLOCK_ROWS)
        tile[threadIdx.y+j][threadIdx.x] = inData[(y+j)*width + x];

    __syncthreads();

    // ===== Transpose the matrix
    x = blockIdx.y * TILE_DIM + threadIdx.x;
    y = blockIdx.x * TILE_DIM + threadIdx.y;

    // ===== Coalesced access to global memory
    for (int j=0; j<TILE_DIM; j+=BLOCK_ROWS)
        outData[(y+j)*width + x] = tile[threadIdx.x][threadIdx.y + j];
}

// ===================== Main =====================
int main(int argc, char *argv[])
{
    // ===== Check arguments
    if (argc != 3)
	{
		printf("./lab5_ex3_seq width height \n");
		return 0;
	}

	// ===== Get width and height
	int width, height;
	width = atoi(argv[1]);
	height = atoi(argv[2]);
    
    // ===== Declare timer variables
	struct timespec start, end;

	// ===== Get total number of elements
	int N = width * height;

    // ===== Show number of elements
	printf("\n-> Dimensions: 		%d x %d\n", width, height);
	printf("-> Number of elements:  %d\n\n", N);

    // ===== Initialize the host matrix
    int aux[N];
    for(int i=0; i<height; i++)
	{
		for (int j=0; j<width; j++)
		{
			aux[i*width + j] = i;
		}
	} 

    // ===== Allocate device memory
    int *device_original = NULL, *device_transposed = NULL, err = 0;
    err = hipMalloc((void **)&device_original, sizeof(int)*N);
    err = hipMalloc((void **)&device_transposed, sizeof(int)*N);
    if(err != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }
    
    // ===== Get initial time
	clock_gettime(CLOCK_MONOTONIC, &start);

    // ===== Copy data from host memory to device memory
    err = hipMemcpy(device_original,aux,sizeof(int)*N,hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }
    
    // ===== Define block and grid dimensions
    dim3 threadsPerBlock(TILE_DIM/4, TILE_DIM/4, 1);
    dim3 blocksPerGrid(width/TILE_DIM, height/TILE_DIM, 1);

    // ===== Launch kernel
    reduce<<<blocksPerGrid,threadsPerBlock>>>(device_original, device_transposed);
    
    // ===== Copy data from device memory to host memory
    int * host_transposed = (int *)malloc(sizeof(int)*N);
    err = hipMemcpy(host_transposed,device_transposed,sizeof(int)*N,hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }
    
    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
	
    // ===== Calculate the elapsed time
    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    
    // ===== Validate results
    printf("host_transposed[%i][%i]=%u\n", width/2, height/4, host_transposed[((width/2) * height) + height/4]);
    printf("host_transposed[%i][%i]=%u\n", width/4, 3*height/4, host_transposed[((width/4) * height) + 3*height/4]);
    printf("host_transposed[%i][%i]=%u\n", 3*width/4, height/2, host_transposed[((3*width/4) * height) + height/2]);
    
    // ===== Print time
	printf("\n--> Execution Time:\t%f ms\n\n", (finalTime - initialTime));

    // ===== Free device buffers
    hipFree(device_original);
    hipFree(device_transposed);

    // ===== Free host buffers
    free(host_transposed);
    
    return 0;
}
