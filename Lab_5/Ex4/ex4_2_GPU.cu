#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex4_2_GPU ex4_2_GPU.cu
// -> Run
// ./ex4_2_GPU
// ============================================================================

#include <stdio.h>
#define bin 16
#define N 65536

// ===================== Kernel =====================
__global__ void histogram(unsigned char * image, int * hist, int size)
{   
    // ===== Shared memory for image
    __shared__ unsigned char s_data[256];

    // ===== Shared memory for histogram
    __shared__ int s_hist[bin];


    // ===== Calculate row and column index
    int Col = threadIdx.x + (blockIdx.x * blockDim.x);
    int Row = threadIdx.y + (blockIdx.y * blockDim.y);

    // ===== Calculate index
    int i = Row * size + Col;
    int i_thread = threadIdx.y * blockDim.x + threadIdx.x;

    // ===== Load image to shared memory
    s_data[i_thread] = image[i];
    if (i_thread < bin)
    {
        s_hist[i_thread] = 0;
    }
    __syncthreads();

    // ===== Update Histogram
    atomicAdd(&s_hist[(int)(s_data[i_thread]/bin)],1);

    __syncthreads();

    if (i_thread < bin)
    {
        atomicAdd(&hist[i_thread + blockIdx.x * bin], s_hist[i_thread]);
    }
}

// ===================== Main =====================

int main()
{
    // ===== Image properties
    int size = 256;

    // ===== Initialize image
    unsigned char aux[N];
    for (int i=0; i<N; i++)
    {
        aux[i] = 255 * sin(i) * sin(i);
    }

    // ===== Allocate device memory
    unsigned char *image = NULL;
    int *hist = NULL;

    int err = 0;
    err = hipMalloc(&image, sizeof(unsigned char)*N);
    err = hipMalloc(&hist, sizeof(int)*bin*bin);
    if(err != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }

    // ===== Declare timer variables
	struct timespec start, end;

    // ===== Get initial time
	clock_gettime(CLOCK_MONOTONIC, &start);

    // ===== Copy data from host memory to device memory
    err = hipMemcpy(image,&aux,sizeof(unsigned char)*N,hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }

    // ===== Define block and grid dimensions
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid(size/16, size/16, 1);

    // ===== Launch kernel
    histogram<<<blocksPerGrid,threadsPerBlock>>>(image, hist, size);

    // ===== Copy data from device memory to host memory
    int * host_image_final = (int *)malloc(sizeof(int)*bin*bin);
    err = hipMemcpy(host_image_final,hist,sizeof(int)*bin*bin,hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }
    
    // ===== Get histogram
    int histogram[bin];
    for (int i=0; i<bin; i++)
    {
        histogram[i] = 0;
        for (int j=0; j<bin; j++)
        {
            histogram[i] += host_image_final[j*bin + i];
        }
    }

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
	
    // ===== Calculate the elapsed time
    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    
    // ===== Validation
    int counter = 0;
	for(int i=0; i<bin; i++)
	{
		printf("hist[%i]=%u\n", i, histogram[i]);
		counter += histogram[i];
	} 

    // ===== Show results
	printf("--> Total pixels = %i\n", counter);
	printf("--> Time (%ix%i elemens):\t%f ms\n", size, size, (finalTime - initialTime));
    
    // ===== Free device memory
    hipFree(image);
    hipFree(hist);

    // ===== Free host memory
    free(host_image_final);

    return 0;
}
