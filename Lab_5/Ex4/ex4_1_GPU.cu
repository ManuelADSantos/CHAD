#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex4_1_GPU ex4_1_GPU.cu
// -> Run
// ./ex4_1_GPU
// ============================================================================

#include <stdio.h>
#define bin 16

// ===================== Kernel =====================
__global__ void histogram(unsigned char * image, int * hist, int size)
{
    // ===== Calculate row and column index
    int Col = threadIdx.x + (blockIdx.x * blockDim.x);
    int Row = threadIdx.y + (blockIdx.y * blockDim.y);

    // ===== Calculate index
    int i = Row * size + Col;
    unsigned char pixel = image[i];
    int index = (int)(pixel/bin);

    // ===== Update histogram
    atomicAdd(&hist[index],1);
}

// ===================== Main =====================
int main()
{   
    // ===== Image properties
    int size = 256;
    int N = size * size;

    // ===== Initialize image
    unsigned char aux[N];
    int aux2[bin];
    for (int i=0; i<N; i++)
    {
        aux[i] = 255 * sin(i) * sin(i);
    }

    for (int i=0; i<bin; i++)
    {
        aux2[i] = 0;
    }

    // ===== Allocate host memory
    int * host_histogram = (int *)malloc(sizeof(int)*bin);
    
    // ===== Allocate device memory
    unsigned char *image = NULL;
    int *hist = NULL;
    int err = 0;
    err = hipMalloc(&image, sizeof(unsigned char)*N);
    err = hipMalloc(&hist, sizeof(int)*bin);
    if(err != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }

    // ===== Declare timer variables
	struct timespec start, end;

    // ===== Get initial time
	clock_gettime(CLOCK_MONOTONIC, &start);

    // ===== Copy data from host memory to device memory
    err = hipMemcpy(image,&aux,sizeof(unsigned char)*N,hipMemcpyHostToDevice);
    err = hipMemcpy(hist,&aux2,sizeof(int)*bin,hipMemcpyHostToDevice);
    if(err != hipSuccess || err != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }

    // ===== Define block and grid dimensions
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid(size/16, size/16, 1);
    
    // ===== Launch kernel
    histogram<<<blocksPerGrid,threadsPerBlock>>>(image, hist, size);

    // ===== Copy data from device memory to host memory
    err = hipMemcpy(host_histogram,hist,sizeof(int)*bin,hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
	
    // ===== Calculate the elapsed time
    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    
    // ===== Validation
    int counter = 0;
	for(int i=0; i<bin; i++)
	{
		printf("hist[%i]=%u\n", i, host_histogram[i]);
		counter += host_histogram[i];
	} 
	
    // ===== Show results
    printf("--> Total pixels = %i\n", counter);
	printf("--> Time (%ix%i elemens):\t%f ms\n", size, size, (finalTime - initialTime));
 
    // ===== Free device memory
    hipFree(image);
    hipFree(hist);

    // ===== Free host memory
    free(host_histogram);

    return 0;
}
