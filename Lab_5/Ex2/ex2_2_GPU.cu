// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_2_GPU ex2_2_GPU.cu -lrt -lm
// ============================================================================


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 1234

// ===================== Kernel ===================== 
__global__ void sum(int *a, int *result)
{
    __shared__ int temp[256];

    //===== Calculate the index of the current thread
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // ===== ADD
    temp[threadIdx.x] = a[index];
    __syncthreads();

    // ===== REDUCE
    for (int i = blockDim.x/2; i > 0; i /= 2)
    {
        if (threadIdx.x < i)
        {
            temp[threadIdx.x] += temp[threadIdx.x + i];
        }
        __syncthreads();
    }

    // ===== Store the result
    if (threadIdx.x == 0)
    {
        atomicAdd(result, temp[0]);
    }
}

// ===================== Main =====================
int main(int argc, char *argv[])
{
    // ===== Show number of elements
    printf("-> Number of elements:  %d\n", N);

    // ===== Declare variables
    int *a, *result;
    int *d_a, *d_result;
    int size = N * sizeof(int);

    // ===== Initialize timer variables
    struct timespec start, end;

    // ===== Allocate host memory
    a = (int*)malloc(size);
    result = (int*)calloc(1, sizeof(int));
    *result = 0;

    // ===== Initialize the vector
    for (int i = 0; i < N; i++) 
    {
        a[i] = i;
    }
    
    // ===== Allocate device memory
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    // ===== Copy data from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    // ===== Initialize the grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = ceil(N/256.0);
    
    // ===== Get initial time
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    // ===== Call the kernel
    sum<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_result);

    // ===== Copy data from device to host
    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);

    // ===== Calculate the elapsed time
    double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
    double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
    printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    // ===== Show the result (validation)
    printf("Sum of all elements in the vector: %d\n", *result);

    // ===== Free device memory
    hipFree(d_a);
    hipFree(d_result);

    // ===== Free host memory
    free(a);
    free(result);

    return 0;
}