// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_1_GPU_normal ex2_1_GPU_normal.cu -lrt -lm
// -> Run
// ./ex2_1_GPU_normal
// ============================================================================

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 1234

__global__ void sum(int* a, int *result)
{
    int tid = threadIdx.x;
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    result[tid] = a[index];
        
    for (int s = 1; s < blockDim.x; s *= 2) {
	if (tid % (2*s) == 0)
    {
	    result[tid] += result[tid + s];
	}
	__syncthreads();
    }
}

int main()
{
    // ===== Show number of elements
    printf("-> Number of elements:  %d\n", N);

    int *a, *result, soma = 0;
    int *d_a, *d_result;
    int size = N * sizeof(int);

    // ===== Initialize timer variables
    struct timespec start, end;

    // ===== Allocate host memory
    a = (int*)malloc(size);
    result = (int*)calloc(N, sizeof(int));
    int check = 0;

    // ===== Initialize vector
    for (int i = 0; i < N; i++) 
    {
        a[i] = i;
    }
    
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256; 
    int blocksPerGrid = (int)ceil(N/256.0);
    
    // ===== Get initial time
    clock_gettime(CLOCK_MONOTONIC, &start);

    sum<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_result);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
    // ===== Calculate the elapsed time
    double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
    double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
    printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    for(int i = 0; i < blocksPerGrid; i++)
    {
    	soma += result[i];
    }
    printf("Sum of all elements in the vector: %d\n", *result);

    
    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
