// nvcc -o ex2_1_GPU_normal ex2_1_GPU_normal.cu -lrt -lm

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void sum(int *a, int *result) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    __syncthreads();
    if (idx < N) {
        result[blockIdx.x] = a[idx];
    }
}

int main() {
    int *a, *result, *d_a, *d_result;
    int size = N * sizeof(int);

    a = (int*)malloc(size);
    result = (int*)calloc(1, sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = i;
        printf("a[%d] = %d | result[%d] = %d\n",i, a[i], i, result[i]);
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(N, 1, 1);
    dim3 dimGrid(1, 1, 1);

    sum<<<dimGrid, dimBlock>>>(d_a, d_result);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum of all elements in the vector: %d\n", *result);

    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
