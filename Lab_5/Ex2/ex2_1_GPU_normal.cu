// nvcc -o ex2_1_GPU_normal ex2_1_GPU_normal.cu -lrt -lm

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

// #define N 100


__global__ void sum(int *a, int *result)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    atomicAdd(result, a[index]);
}

int main(int argc, char *argv[])
{
    // Check if the number of arguments is correct
    if (argc != 2)
    {
        printf("./ex2_CPU num_elements\n");
        return -1;
    }
    int N = atoi(argv[1]);
    printf("Number of elements: %d\n", N);

    int *a, *result;
    int *d_a, *d_result;
    int size = N * sizeof(int);

    // ===== Initialize timer variables
    struct timespec start, end;


    a = (int*)malloc(size);
    result = (int*)calloc(1, sizeof(int));
    *result = 0;

    //printf("result = %d\n", *result);
    for (int i = 0; i < N; i++) 
    {
        a[i] = i;
        //printf("a[%d] = %d  ",i, a[i]);
    }
    
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = ceil(N/256.0);
    
    // ===== Get initial time
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    sum<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_result);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
    // ===== Calculate the elapsed time
    double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
    double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
    printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    printf("Sum of all elements in the vector: %d\n", *result);

    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
