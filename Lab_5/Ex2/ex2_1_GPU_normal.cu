// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_1_GPU_normal ex2_1_GPU_normal.cu -lrt -lm

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 100


__global__ void sum(int *a, int *result)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    atomicAdd(result, a[index]);
}

int main()
{
    int *a, *result;
    int *d_a, *d_result;
    int size = N * sizeof(int);

    // ===== Initialize timer variables
    struct timespec start, end;


    a = (int*)malloc(size);
    result = (int*)calloc(1, sizeof(int));
    *result = 0;

    //printf("result = %d\n", *result);
    for (int i = 0; i < N; i++) 
    {
        a[i] = i;
        //printf("a[%d] = %d  ",i, a[i]);
    }
    
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(N, 1, 1);
    dim3 dimGrid(1, 1, 1);
    
    // ===== Get initial time
    clock_gettime(CLOCK_MONOTONIC, &start);

    sum<<<dimGrid, dimBlock>>>(d_a, d_result);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
    // ===== Calculate the elapsed time
    double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
    double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
    printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    printf("Sum of all elements in the vector: %d\n", *result);

    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
