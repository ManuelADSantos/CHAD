// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_1_GPU_normal ex2_1_GPU_normal.cu -lrt -lm
// -> Run
// ./ex2_1_GPU_normal
// ============================================================================

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// ===================== Kernel function =====================
__global__ void sum(int *a, int *result)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    atomicAdd(result, a[index]);
}

// ===================== Main function =====================
int main(int argc, char *argv[])
{
    // ===== Check if the number of arguments is correct
    if (argc != 2)
    {
        printf("./ex2_1_GPU_normal num_elements\n");
        return -1;
    }

    // ===== Get the number of elements
    int N = atoi(argv[1]);
    printf("-> Number of elements:  %d\n", N);

    // ===== Initialize variables
    int *a, *result;
    int *d_a, *d_result;
    int size = N * sizeof(int);

    // ===== Initialize timer variables
    struct timespec start, end;

    // ===== Allocate memory
    a = (int*)malloc(size);
    result = (int*)calloc(1, sizeof(int));
    *result = 0;

    // ===== Initialize vector
    for (int i = 0; i < N; i++) 
    {
        a[i] = i;
    }
    
    // ===== Allocate memory on the device
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    // ===== Initialize the grid and block dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = ceil(N/256.0);
    
    // ===== Get initial time
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    // ===== Copy data from host to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    // ===== Execute the kernel
    sum<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_result);

    // ===== Copy data from device to host
    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);

    // ===== Calculate the elapsed time
    double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
    double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
    printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    // ===== Print the result (Validating the result)
    printf("Sum of all elements in the vector: %d\n", *result);

    // ===== Free device memory
    hipFree(d_a);
    hipFree(d_result);

    // ===== Free host memory
    free(a);
    free(result);

    return 0;
}
