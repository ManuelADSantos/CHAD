// nvcc -o ex2_1_GPU_normal ex2_1_GPU_normal.cu -lrt -lm

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10


__global__ void sum(int *a, int *result)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    atomicAdd(result, a[index]);
}

int main()
{
    int *a, *result;
    int *d_a, *d_result;
    int size = N * sizeof(int);

    a = (int*)malloc(size);
    result = (int*)calloc(1, sizeof(int));
    *result = 0;

    printf("result = %d\n", *result);
    for (int i = 0; i < N; i++) 
    {
        a[i] = i;
        printf("a[%d] = %d  ",i, a[i]);
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(N, 1, 1);
    dim3 dimGrid(1, 1, 1);

    sum<<<dimGrid, dimBlock>>>(d_a, d_result);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("\nSum of all elements in the vector: %d\n", *result);

    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
