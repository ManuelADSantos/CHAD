#include "hip/hip_runtime.h"
// nvcc -o lab5_ex2_2 lab5_ex2_2.cu
#include <stdio.h>
#define N 9984

// Device code
__global__ void reduce(int *g_idata, int *g_odata)
{
    __shared__ int s_data[N];
    unsigned int tid = threadIdx.x;
    unsigned int gid = threadIdx.x + (blockIdx.x * blockDim.x); 

    s_data[tid] = g_idata[gid];
    __syncthreads();

    //reduction in shared memory
    for(unsigned int s=1; s<blockDim.x; s*=2)
    {
        if (tid % (2*s) == 0)
        {
            
            s_data[tid] += s_data[tid+s];
        }
        __syncthreads();
    }

    //write result to global memory
    if (tid == 0)
        g_odata[blockIdx.x] = s_data[tid];
}

int main()
{
    //inicializa��o do vetor
    int aux[N];
    int final = 0;
    for (int i=0; i<N; i++)
    {
        aux[i] = 1;
    }
    int *v1 = NULL;
    int *v2 = NULL;
    int err1 = 0;
    int err2 = 0;
    err1 = hipMalloc(&v1, sizeof(int)*N);
    err2 = hipMalloc(&v2, sizeof(int)*(N/256));
    if(err1 != hipSuccess || err2 != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }
    //CLOCK_PROCESS_CPUTIME_ID - Profiling the execution time of loop
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);

    err1 = hipMemcpy(v1,&aux,sizeof(int)*N,hipMemcpyHostToDevice);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }
    // Launch device function
    int threadsPerBlock = 256;
    int blocksPerGrid = N/256;
    reduce<<<blocksPerGrid,threadsPerBlock>>>(v1, v2);
    
    // Copy data from device memory to host memory
    int * host_buffer = (int *)malloc(sizeof(int)*(N/256));
    err1 = hipMemcpy(host_buffer,v2,sizeof(int)*(N/256),hipMemcpyDeviceToHost);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }
    for (int i=0; i<N/256; i++)
    {
        final += host_buffer[i];
    }
    clock_gettime(CLOCK_MONOTONIC, &end);
	double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    printf("final = %i\n", final);
    printf("Cuda (%i elemens):\t%f ms\n", N, (finalTime - initialTime));
    // Free device buffers
    hipFree(v1);
    hipFree(v2);
    return 0;
}
