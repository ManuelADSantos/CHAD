// nvcc -o ex2_2_GPU_optim ex2_2_GPU_optim.cu -lrt -lm

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define N 100

__global__ void sum(int *a, int *result) {
    __shared__ int temp[N];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    temp[tid] = a[i];
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) 
        {
            temp[tid] += temp[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        *result = temp[0];
    }
}

int main() {
    int *a, *result, *d_a, *d_result;
    int size = N * sizeof(int);

    // ===== Initialize timer variables
    struct timespec start, end;

    a = (int*)malloc(size);
    result = (int*)malloc(sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = i;
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(N, 1, 1);
    dim3 dimGrid(1, 1, 1);

    // ===== Get initial time
    clock_gettime(CLOCK_MONOTONIC, &start);

    sum<<<dimGrid, dimBlock>>>(d_a, d_result);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
    // ===== Calculate the elapsed time
    double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
    double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
    printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    printf("Sum of all elements in the vector: %d\n", *result);

    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
