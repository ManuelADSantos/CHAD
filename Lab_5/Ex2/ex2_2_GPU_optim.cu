#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_2_GPU_optim ex2_2_GPU_optim.cu -lrt -lm
// -> Run
// ./ex2_2_GPU_optim
// ============================================================================
#include <stdio.h>
#include <time.h>

#define N 10000

__global__ void sum(int *a, int *result)
{
    __shared__ int temp[N];

    int tid = threadIdx.x;
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    // Load elements into shared memory
    soma[tid] = v1[index];
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) 
        {
            temp[tid] += temp[tid + s];
        }
        __syncthreads();
    }

    // Result in global memory
    if(tid == 0){
	result[blockIdx.x] = soma[tid];
    }
}

int main(int argc, char *argv[])
{
    // Check if the number of arguments is correct
    // if (argc != 2)
    // {
    //     printf("./ex2_CPU num_elements\n");
    //     return -1;
    // }
    //int N = atoi(argv[1]);
    printf("Number of elements: %d\n", N);

    int *a, *result, *d_a, *d_result;
    int size = N * sizeof(int);

    // ===== Initialize timer variables
    struct timespec start, end;

    a = (int*)malloc(size);
    result = (int*)malloc(sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = i;
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = ceil(N/256.0);

    // ===== Get initial time
    clock_gettime(CLOCK_MONOTONIC, &start);

    sum<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_result);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
    // ===== Calculate the elapsed time
    double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
    double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
    printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    printf("Sum of all elements in the vector: %d\n", *result);

    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
