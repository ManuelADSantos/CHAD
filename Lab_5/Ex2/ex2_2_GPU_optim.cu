#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_2_GPU_optim ex2_2_GPU_optim.cu -lrt -lm
// -> Run
// ./ex2_2_GPU_optim
// ============================================================================
#include <stdio.h>
#include <time.h>

// ===================== Kernel function =====================
__global__ void sum(int* input, int* output, int N) {
    __shared__ int soma[N]; // Allocate shared memory
    int tid = threadIdx.x;
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    // Load elements into shared memory
    soma[tid] = v1[index];
    __syncthreads();
        
    // Reduction in shared memory
    for (int s = 1; s < blockDim.x; s *= 2) {
	if (tid % (2*s) == 0) {
	    soma[tid] += soma[tid + s];
		
	}
	__syncthreads();
    }

    // Result in global memory
    if(tid == 0){
	result[blockIdx.x] = soma[tid];
    }
}


// ===================== Main function =====================
int main(int argc, char *argv[])
{
    // Check if the number of arguments is correct
    if (argc != 2)
    {
        printf("./ex2_2_GPU_optim num_elements\n");
        return -1;
    }
    int N = atoi(argv[1]);
    printf("-> Number of elements:  %d\n", N);

    int *a, *result, *d_a, *d_result;
    int size = N * sizeof(int);

    // ===== Initialize timer variables
    struct timespec start, end;

    a = (int*)malloc(size);
    result = (int*)malloc(sizeof(int));

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = ceil(N/256.0);
    int sharedMemorySize = N * sizeof(int);

    // ===== Get initial time
    clock_gettime(CLOCK_MONOTONIC, &start);

    sum<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(d_a, d_result, N);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
    // ===== Calculate the elapsed time
    double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
    double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
    printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    printf("Sum of all elements in the vector: %d\n", *result);

    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
