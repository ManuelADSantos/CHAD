// nvcc -o ex2_2_GPU_optim ex2_2_GPU_optim.cu -lrt -lm

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void sum(int *a, int *result) {
    __shared__ int temp[N];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    temp[tid] = a[i];
    __syncthreads();

    for (int s = 1; s < blockDim.x; s *= 2) {
        if (tid % (2 * s) == 0) {
            temp[tid] += temp[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        *result = temp[0];
    }
}

int main() {
    int *a, *result, *d_a, *d_result;
    int size = N * sizeof(int);

    a = (int*)malloc(size);
    result = (int*)malloc(sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = i;
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(N, 1, 1);
    dim3 dimGrid(1, 1, 1);

    sum<<<dimGrid, dimBlock>>>(d_a, d_result);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum of all elements in the vector: %d\n", *result);

    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
