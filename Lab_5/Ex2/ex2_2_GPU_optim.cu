// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 03/11/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_2_GPU_optim ex2_2_GPU_optim.cu -lrt -lm
// -> Run
// ./ex2_2_GPU_optim num_elements
// ============================================================================

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

// ===================== Kernel function =====================
__global__ void sum(int *a, int *result, int n)
{
    extern __shared__ int temp[];

    int tid = threadIdx.x;
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    // Initialize temp array with input or 0 if out of bounds
    temp[tid] = (i < n) ? a[i] : 0;
    __syncthreads();

    for (int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) 
        {
            temp[tid] += temp[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        atomicAdd(result, temp[0]);
    }
}

// ===================== Main function =====================
int main(int argc, char *argv[])
{
    // Check if the number of arguments is correct
    if (argc != 2)
    {
        printf("./ex2_2_GPU_optim num_elements\n");
        return -1;
    }
    int N = atoi(argv[1]);
    printf("-> Number of elements:  %d\n", N);

    int *a, *result, *d_a, *d_result;
    int size = N * sizeof(int);

    // ===== Initialize timer variables
    struct timespec start, end;

    a = (int*)malloc(size);
    result = (int*)malloc(sizeof(int));

    for (int i = 0; i < N; i++)
    {
        a[i] = i;
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = ceil(N/256.0);
    int sharedMemorySize = N * sizeof(int);

    // ===== Get initial time
    clock_gettime(CLOCK_MONOTONIC, &start);

    sum<<<blocksPerGrid, threadsPerBlock, sharedMemorySize>>>(d_a, d_result, N);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    // ===== Get final time
    clock_gettime(CLOCK_MONOTONIC, &end);
    // ===== Calculate the elapsed time
    double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
    double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
    printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    printf("Sum of all elements in the vector: %d\n", *result);

    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
