// nvcc -o ex2_1_GPU_optim ex2_1_GPU_optim.cu -lrt -lm

#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 10

__global__ void sum(int *a, int *result) {
    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    __shared__ int sdata[N];

    sdata[tid] = a[idx];
    __syncthreads();

    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    if (tid == 0) {
        result[blockIdx.x] = sdata[0];
    }
}

int main() {
    int *a, *result, *d_a, *d_result;
    int size = N * sizeof(int);

    a = (int*)malloc(size);
    result = (int*)malloc(sizeof(int));

    for (int i = 0; i < N; i++) {
        a[i] = i;
    }

    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_result, sizeof(int));

    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);

    dim3 dimBlock(N, 1, 1);
    dim3 dimGrid(1, 1, 1);

    sum<<<dimGrid, dimBlock>>>(d_a, d_result);

    hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);

    printf("Sum of all elements in the vector: %d\n", *result);

    hipFree(d_a);
    hipFree(d_result);
    free(a);
    free(result);

    return 0;
}
