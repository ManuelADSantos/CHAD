#include "hip/hip_runtime.h"
// nvcc -o lab5_ex5_2 lab5_ex5_2.cu
#include <stdio.h>
#define TILE 16

// Device code
__global__ void reduce(int *m1, int *m2, int *m3, int width, int height)
{
    __shared__ int sub1[TILE][TILE];
    __shared__ int sub2[TILE][TILE];

    int bx = blockIdx.x;
    int by = blockIdx.y;
    int tx = threadIdx.x;
    int ty = threadIdx.y;

    int col = tx + bx * TILE;
    int row = ty + by * TILE;
    int vfinal = 0;

    // Loop que corre todos os sub-blocos necess�rios para calcular 1 elemento
    for (int m=0; m<width/TILE; ++m)
    {
        sub1[ty][tx] = m1[row*width + m*TILE + tx];
        sub2[ty][tx] = m2[(m*TILE + ty)*width + col];
        __syncthreads();
        for (int k=0; k<TILE; ++k)
            vfinal += sub1[ty][k] * sub2[k][tx];
        __syncthreads();
    }

    m3[row * width + col] = vfinal;
}

int main()
{
    int width = 256;
    int height = 256;
    int N = width * height;

    //inicializa��o do vetor
    int aux1[N];
    int aux2[N];
    for(int i=0; i<N; i++)
	{
		aux1[i] = 1;
        aux2[i] = 2;
	} 

    int *m1 = NULL;
    int *m2 = NULL;
    int *m3 = NULL;
    int err1 = 0;
    int err2 = 0;
    int err3 = 0;
    err1 = hipMalloc(&m1, sizeof(int)*N);
    err2 = hipMalloc(&m2, sizeof(int)*N);
    err3 = hipMalloc(&m3, sizeof(int)*N);
    if(err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }
    //CLOCK_PROCESS_CPUTIME_ID - Profiling the execution time of loop
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);

    err1 = hipMemcpy(m1,&aux1,sizeof(int)*N,hipMemcpyHostToDevice);
    err2 = hipMemcpy(m2,&aux2,sizeof(int)*N,hipMemcpyHostToDevice);
    if(err1 != hipSuccess || err2 != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }
    // Launch device function
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid(width/16, height/16, 1);
    reduce<<<blocksPerGrid,threadsPerBlock>>>(m1, m2, m3, width, height);
    
    // Copy data from device memory to host memory
    int * host_buffer = (int *)malloc(sizeof(int)*N);
    err1 = hipMemcpy(host_buffer,m3,sizeof(int)*N,hipMemcpyDeviceToHost);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }
    clock_gettime(CLOCK_MONOTONIC, &end);
	double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    printf("host_buffer[%i][%i]=%u\n", width/2, height/4, host_buffer[((width/2) * height) + height/4]);
    printf("host_buffer[%i][%i]=%u\n", width/4, 3*height/4, host_buffer[((width/4) * height) + 3*height/4]);
    printf("host_buffer[%i][%i]=%u\n", 3*width/4, height/2, host_buffer[((3*width/4) * height) + height/2]);
    printf("Cuda (%ix%i elemens):\t%f ms\n", width, height, (finalTime - initialTime));
    // Free device buffers
    hipFree(m1);
    hipFree(m2);
    hipFree(m3);
    return 0;
}
