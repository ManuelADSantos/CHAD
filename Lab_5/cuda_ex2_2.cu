﻿// nvcc -o cuda_ex2_2 cuda_ex2_2.cu

#include <hip/hip_runtime.h>
#include <stdio.h>  
#include <stdlib.h> 
#include <math.h> 
#include <time.h> 

#define N 1234

__global__ void somaCudaOtimizado(int* v1, int *result)
{
	
    __shared__ int soma[N]; // Allocate shared memory
    int tid = threadIdx.x;
    int index = threadIdx.x + (blockIdx.x * blockDim.x);

    // Load elements into shared memory
    soma[tid] = v1[index];
    __syncthreads();
        
    // Reduction in shared memory
    for (int s = 1; s < blockDim.x; s *= 2) {
	if (tid % (2*s) == 0) {
	    soma[tid] += soma[tid + s];
		
	}
	__syncthreads();
    }

    // Result in global memory
    if(tid == 0){
	result[blockIdx.x] = soma[tid];
    }
}


int main( int argc, char** argv){

    //Criar Imagem com valores random
    //int N = 2048;
    int result = 0;
    int result2 = 0;
    int *v1 = (int*)malloc(N * sizeof(int));
    int *vOut = (int*)malloc(N * sizeof(int));

    for (int i = 0; i < N; i++){
	v1[i] = i;
	vOut[i] = 0;
    }


    // Versão sequencial
    for (int i = 0; i < N; i++){
	result += v1[i];
    }
	
    printf("Resultado da Soma - Sequencial: %d \n", result);
    
    
    // Versão paralela
    
    int threadsPerBlock = 256; // 256
    int blocksPerGrid = N/256;

    int *v1Cuda = NULL;
    int *vOutCuda = NULL;

    hipMalloc(&v1Cuda, N * sizeof(int));
    hipMalloc(&vOutCuda, N * sizeof(int));

    hipMemcpy(v1Cuda, v1, N * sizeof(int), hipMemcpyHostToDevice);
    
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);

    somaCudaOtimizado<<<blocksPerGrid, threadsPerBlock>>>(v1Cuda, vOutCuda);
    
    clock_gettime(CLOCK_MONOTONIC, &end);
    
    hipMemcpy(vOut, vOutCuda, N * sizeof(int), hipMemcpyDeviceToHost);
    
    //clock_gettime(CLOCK_MONOTONIC, &end);
    
    for(int i = 0; i < blocksPerGrid; i++){
    	result2 += vOut[i];
    }
	
    //clock_gettime(CLOCK_MONOTONIC, &end);

    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
    double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    
    
    
    printf("Exercício 2.2 - CUDA com otimizações: %f ms\n", (finalTime - initialTime));
    printf("Resultado da Soma - CUDA com otimizações: %d \n", result2);
    /*
    cilo for até blocksPerGrid
    
    if(vOut == result)
        printf("A soma do vetor foi bem sucedida.\n");
    else
        printf("A soma do vetor não foi bem sucedida.\n");
*/

    hipFree(v1Cuda);
    hipFree(vOutCuda);
    
    free(v1);
    free(vOut);

    return 0;

}

