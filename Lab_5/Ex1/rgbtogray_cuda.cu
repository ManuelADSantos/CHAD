// nvcc -o rgbtogray_cuda rgbtogray_cuda.cu -lrt -lm


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#define CHANNELS 3 // we have 3 channels corresponding to RGB

typedef struct
{
    unsigned char red, green, blue;
} PPMPixel;

typedef struct
{
    int x, y;
    int color_value;
    PPMPixel *data;
} PPMImage;

// Device code
__global__ void colorToGreyScaleConvertion( PPMPixel *grayImage, PPMPixel *rgbImage, int width, int height)
{
    int Col = threadIdx.x + (blockIdx.x * blockDim.x);
    int Row = threadIdx.y + (blockIdx.y * blockDim.y);
    if (Col < width && Row < height)
    {
        // get 1D coordinate for the grayscale image
        int greyOffset = Row * width + Col;
        // one can think of the RGB image having
        // CHANNEL times columns of the gray scale image
        // int rgbOffset = greyOffset * CHANNELS;
        int rgbOffset = greyOffset;
        unsigned char r = rgbImage[rgbOffset].red;
        // red value for pixel
        unsigned char g = rgbImage[rgbOffset].green;
        // green value for pixel
        unsigned char b = rgbImage[rgbOffset].blue;
        // blue value for pixel
        // perform the rescaling and store it
        // We multiply by floating point constants
        grayImage[greyOffset].red = grayImage[greyOffset].green = grayImage[greyOffset].blue = 0.21f * r + 0.71f * g + 0.07f * b;
        // grayImage[greyOffset].red = r;
        // grayImage[greyOffset].green = g;
        // grayImage[greyOffset].blue = b;
    }
}
// ===========================================================

static PPMImage *readPPM(const char *filename)
{
    char buff[16];
    PPMImage *img;
    FILE *fp;
    // open PPM file for reading
    fp = fopen(filename, "rb");
    if (!fp)
    {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    // read image format
    if (!fgets(buff, sizeof(buff), fp))
    {
        perror(filename);
        exit(1);
    }

    // check the image format
    if (buff[0] != 'P' || buff[1] != '6')
    {
        fprintf(stderr, "Invalid image format (must be 'P6')\n");
        exit(1);
    }

    // alloc memory form image
    img = (PPMImage *)malloc(sizeof(PPMImage));
    if (!img)
    {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    // read image size information
    if (fscanf(fp, "%d %d", &img->x, &img->y) != 2)
    {
        fprintf(stderr, "Invalid image size (error loading '%s')\n", filename);
        exit(1);
    }

    // read rgb component
    if (fscanf(fp, "%d", &img->color_value) != 1)
    {
        fprintf(stderr, "Invalid rgb component (error loading '%s')\n", filename);
        exit(1);
    }

    while (fgetc(fp) != '\n')
        ;

    // memory allocation for pixel data
    img->data = (PPMPixel *)malloc(img->x * img->y * sizeof(PPMPixel));

    if (!img)
    {
        fprintf(stderr, "Unable to allocate memory\n");
        exit(1);
    }

    // read pixel data from file
    if (fread(img->data, 3 * img->x, img->y, fp) != img->y)
    {
        fprintf(stderr, "Error loading image '%s'\n", filename);
        exit(1);
    }

    fclose(fp);
    return img;
}
void writePPM(const char *filename, PPMImage *img)
{
    FILE *fp;
    // open file for output
    fp = fopen(filename, "wb");
    if (!fp)
    {
        fprintf(stderr, "Unable to open file '%s'\n", filename);
        exit(1);
    }

    // write the header file
    // image format
    fprintf(fp, "P6\n");
    // image size
    fprintf(fp, "%d\n%d\n", img->x, img->y);

    // rgb component depth
    fprintf(fp, "%d\n", img->color_value);

    // pixel data
    fwrite(img->data, 3 * img->x, img->y, fp);

    fclose(fp);
}

int main(int argc, char *argv[])
{
    // int height = 5184, width = 3456;
    int err;

    if (argc == 2)
    {
        PPMImage *image, *gs_image;
        PPMPixel *d_image_data, *d_gs_image_data;
        image = readPPM(argv[1]);

        int height = image->x, width = image->y;

        // for (int i = 0; i < 20; i++)
        // {
        //     printf("Pixel %d: %d %d %d\n", i, image->data[i].red, image->data[i].green, image->data[i].blue);
        // }

        // ===== Initialize timer variables
        struct timespec start, end;

        gs_image = (PPMImage *)malloc(sizeof(PPMImage));
        gs_image->data = (PPMPixel *)malloc(image->x * image->y * sizeof(PPMPixel));
        
        gs_image->color_value = image->color_value;
        // printf("Color Value: %d\n", image->color_value);
        // printf("GS Color Value: %d\n", gs_image->color_value);

        gs_image->x = image->x;
        // printf("GS X: %d\n", gs_image->x);
        // printf("X: %d\n", image->x);

        gs_image->y = image->y;
        // printf("GS Y: %d\n", gs_image->y);
        // printf("Y: %d\n", image->y);
        
        // ===== Allocate memory for the device (rgb image)
        err = hipMalloc(&d_image_data, sizeof(PPMPixel) * image->x * image->y);
        if(err != hipSuccess)
        {
            perror("Error: hipMalloc &d_image_data\n");
            return -1;
        }
        // ===== Copy the host (rgb image) to the device
        err = hipMemcpy(d_image_data, image->data, sizeof(PPMPixel) * image->x * image->y, hipMemcpyHostToDevice);
        if(err != hipSuccess)
        {
            perror("Error: hipMemcpy image->data to d_image_data (H2D)\n");
            return -1;
        }
        
        // ===== Allocate memory for the device (gs image)
        err = hipMalloc(&d_gs_image_data, sizeof(PPMPixel) * image->x * image->y);
        if(err != hipSuccess)
        {
            perror("Error: hipMalloc &d_gs_image_data\n");
            return -1;
        }

        // ===== Get initial time
        clock_gettime(CLOCK_MONOTONIC, &start);
        colorToGreyScaleConvertion<<<ceil(width / 32.0), ceil(height / 32.0)>>>(d_gs_image_data, d_image_data, width, height);
        hipDeviceSynchronize();
        
        // ===== Copy the result back to the host
        err = hipMemcpy(gs_image->data, d_gs_image_data, sizeof(PPMPixel) * image->x * image->y, hipMemcpyDeviceToHost);
        if(err != hipSuccess)
        {
            perror("Error: hipMemcpy d_gs_image_data to gs_image->data (D2H)\n");
            return -1;
        }

        // ===== Get final time
        clock_gettime(CLOCK_MONOTONIC, &end);
        // ===== Calculate the elapsed time
        double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
        double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
        printf("-> GPU Execution Time:\t%f ms\n", (finalTime - initialTime));
        
        for (int i = 0; i < 10; i++)
        {
            printf("Pixel %d: %d %d %d\n", i, gs_image->data[i].red, gs_image->data[i].green, gs_image->data[i].blue);
        }

        writePPM("output_GPU.ppm", gs_image);

        writePPM("output_GPU_original.ppm", image);

        free(gs_image->data);
        free(gs_image);
        free(image->data);
        free(image);
        hipFree(d_image_data);
        hipFree(d_gs_image_data);
    }
    else
    {
        printf("Usage: ./rgbtogray_cuda <input_image>\n");
        return -1;
    }

    return 0;
}