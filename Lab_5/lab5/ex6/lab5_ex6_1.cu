#include "hip/hip_runtime.h"
// nvcc -o lab5_ex6_1 lab5_ex6_1.cu
#include <stdio.h>
#define t_filtro 3
#define N 65536

// Device code
__global__ void convolution(unsigned char *img, unsigned char *res, float *filtro, int size)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    int aux[t_filtro*t_filtro];
    float valor = 0;

    for (int i=0; i<t_filtro; i++)
    {
        for (int j=0; j<t_filtro; j++)
        {
            if(row+i-1 < 0 || row+i-1 == size || col+j-1 < 0 || col+j-1 == size)
				aux[i*t_filtro + j] = 0;
			else
				aux[i*t_filtro + j] = img[(row+i-1)*size + (col+j-1)];
        }
    }
    for (int n=0; n<t_filtro*t_filtro; n++)
	{
		valor += aux[n] * filtro[n]; 
	}
	res[row*size + col] = (unsigned char)valor;
}

int main()
{
    int size = 256;

    //inicializa��o do vetor
    unsigned char aux1[N];
    float aux2[t_filtro*t_filtro];
    for(int i=0; i<size; i++)
	{
		for(int j=0; j<size; j++)
  		{
  			if (i%2 == j%2)
                aux1[i*size + j] = 255;
  			else 
                aux1[i*size + j] = 0;
  		} 
	} 

    for(int i=0; i<t_filtro*t_filtro; i++)
	{
		aux2[i] = 1;
		aux2[i] = aux2[i]/9;
	} 

    unsigned char *img = NULL;
    float *filtro = NULL;
    unsigned char *res = NULL;
    int err1 = 0;
    int err2 = 0;
    int err3 = 0;
    err1 = hipMalloc(&img, sizeof(unsigned char)*N);
    err2 = hipMalloc(&filtro, sizeof(float)*t_filtro*t_filtro);
    err3 = hipMalloc(&res, sizeof(unsigned char)*N);
    if(err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }
    //CLOCK_PROCESS_CPUTIME_ID - Profiling the execution time of loop
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);

    err1 = hipMemcpy(img,&aux1,sizeof(unsigned char)*N,hipMemcpyHostToDevice);
    err2 = hipMemcpy(filtro,&aux2,sizeof(float)*t_filtro*t_filtro,hipMemcpyHostToDevice);
    if(err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }
    // Launch device function
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid(size/16, size/16, 1);
    convolution<<<blocksPerGrid,threadsPerBlock>>>(img, res, filtro, size);
    
    // Copy data from device memory to host memory
    unsigned char * host_buffer = (unsigned char *)malloc(sizeof(unsigned char)*N);
    err1 = hipMemcpy(host_buffer,res,sizeof(unsigned char)*N,hipMemcpyDeviceToHost);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }
    clock_gettime(CLOCK_MONOTONIC, &end);
	double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    for(int i=0; i<10; i++)
	{
		for(int j=0; j<10; j++)
		{
			printf("%-4u ", host_buffer[i*size + j]);
		} 
		printf("\n");
	} 
    printf("Cuda (%ix%i elemens):\t%f ms\n", size, size, (finalTime - initialTime));
    // Free device buffers
    hipFree(img);
    hipFree(filtro);
    hipFree(res);
    return 0;
}
