#include "hip/hip_runtime.h"
// nvcc -o lab5_ex6_2 lab5_ex6_2.cu
#include <stdio.h>
#define t_filtro 3
#define N 65536
#define TILE 16

// Device code
__global__ void convolution(unsigned char *img, unsigned char *res, float *filtro, int size)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    int tcol = threadIdx.x;
    int trow = threadIdx.y;

    float valor = 0;

    __shared__ unsigned char image[TILE][TILE];
    __shared__ float fil[t_filtro][t_filtro];

    if (col < 3 && row < 3)
    {
        fil[row][col] = filtro[row*3 + col];
    }

    image[trow][tcol] = img[row*size + col];

    __syncthreads();

    for (int i=0; i<t_filtro; i++)
    {
        for (int j=0; j<t_filtro; j++)
        {
            if ((trow + i - 1)<0  || (trow + i - 1)>15 || (tcol + j - 1)<0 || (tcol + j - 1)>15)
            {
                if ((row + i - 1)>=0  || (row + i - 1)<=255 || (col + j - 1)>=0 || (col + j - 1)<=255)
                    valor += img[(row + i - 1)*size + (col + j - 1)];
            }
            else
                valor += image[trow + i - 1][tcol + j - 1] * fil[i][j]; 
        }
    }
    res[row*size + col] = (unsigned char)valor;
}

int main()
{
    int size = 256;

    //inicializa��o do vetor
    unsigned char aux1[N];
    float aux2[t_filtro*t_filtro];
    for(int i=0; i<size; i++)
	{
		for(int j=0; j<size; j++)
  		{
  			if (i%2 == j%2)
                aux1[i*size + j] = 255;
  			else 
                aux1[i*size + j] = 0;
  		} 
	} 

    for(int i=0; i<t_filtro*t_filtro; i++)
	{
		aux2[i] = 1;
		aux2[i] = aux2[i]/9;
	} 

    unsigned char *img = NULL;
    float *filtro = NULL;
    unsigned char *res = NULL;
    int err1 = 0;
    int err2 = 0;
    int err3 = 0;
    err1 = hipMalloc(&img, sizeof(unsigned char)*N);
    err2 = hipMalloc(&filtro, sizeof(float)*t_filtro*t_filtro);
    err3 = hipMalloc(&res, sizeof(unsigned char)*N);
    if(err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }
    //CLOCK_PROCESS_CPUTIME_ID - Profiling the execution time of loop
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);

    err1 = hipMemcpy(img,&aux1,sizeof(unsigned char)*N,hipMemcpyHostToDevice);
    err2 = hipMemcpy(filtro,&aux2,sizeof(float)*t_filtro*t_filtro,hipMemcpyHostToDevice);
    if(err1 != hipSuccess || err2 != hipSuccess || err3 != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }
    // Launch device function
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid(size/16, size/16, 1);
    convolution<<<blocksPerGrid,threadsPerBlock>>>(img, res, filtro, size);
    
    // Copy data from device memory to host memory
    unsigned char * host_buffer = (unsigned char *)malloc(sizeof(unsigned char)*N);
    err1 = hipMemcpy(host_buffer,res,sizeof(unsigned char)*N,hipMemcpyDeviceToHost);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }
    clock_gettime(CLOCK_MONOTONIC, &end);
	double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    for(int i=0; i<10; i++)
	{
		for(int j=0; j<10; j++)
		{
			printf("%-4u ", host_buffer[i*size + j]);
		} 
		printf("\n");
	} 
    printf("Cuda (%ix%i elemens):\t%f ms\n", size, size, (finalTime - initialTime));
    // Free device buffers
    hipFree(img);
    hipFree(filtro);
    hipFree(res);
    return 0;
}
