#include "hip/hip_runtime.h"
// nvcc -o lab5_ex4_2 lab5_ex4_2.cu
#include <stdio.h>
#define bin 16
#define N 65536

// Device code
__global__ void histogram(unsigned char * image, int * hist, int size)
{
    __shared__ unsigned char s_data[256];
    __shared__ int s_hist[bin];

    int Col = threadIdx.x + (blockIdx.x * blockDim.x);
    int Row = threadIdx.y + (blockIdx.y * blockDim.y);

    int i = Row * size + Col;
    int i_thread = threadIdx.y * blockDim.x + threadIdx.x;

    s_data[i_thread] = image[i];
    if (i_thread < bin)
    {
        s_hist[i_thread] = 0;
    }
    __syncthreads();

    atomicAdd(&s_hist[(int)(s_data[i_thread]/bin)],1);

    __syncthreads();

    if (i_thread < bin)
    {
        atomicAdd(&hist[i_thread + blockIdx.x * bin], s_hist[i_thread]);
    }
}

int main()
{
    int size = 256;

    //inicializa��o da imagem
    unsigned char aux[N];
    for (int i=0; i<N; i++)
    {
        aux[i] = 255 * sin(i) * sin(i);
    }

    //aloca��o de mem�ria para a GPU
    unsigned char *image = NULL;
    int *hist = NULL;

    int err1 = 0;
    int err2 = 0;
    err1 = hipMalloc(&image, sizeof(unsigned char)*N);
    err2 = hipMalloc(&hist, sizeof(int)*bin*bin);
    if(err1 != hipSuccess || err2 != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }

    //CLOCK_PROCESS_CPUTIME_ID - Profiling the execution time of loop
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);

    err1 = hipMemcpy(image,&aux,sizeof(unsigned char)*N,hipMemcpyHostToDevice);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }

    // Launch device function
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid(size/16, size/16, 1);
    histogram<<<blocksPerGrid,threadsPerBlock>>>(image, hist, size);

    // Copy data from device memory to host memory
    int * host_buffer = (int *)malloc(sizeof(int)*bin*bin);
    err1 = hipMemcpy(host_buffer,hist,sizeof(int)*bin*bin,hipMemcpyDeviceToHost);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }
    int histograma[bin];
    for (int i=0; i<bin; i++)
    {
        histograma[i] = 0;
        for (int j=0; j<bin; j++)
        {
            histograma[i] += host_buffer[j*bin + i];
        }
    }

    clock_gettime(CLOCK_MONOTONIC, &end);
	double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    int contador = 0;
	for(int i=0; i<bin; i++)
	{
		printf("hist[%i]=%u\n", i, histograma[i]);
		contador += histograma[i];
	} 
	printf("Contagem total = %i\n", contador);
    printf("Cuda (%i*%i elemens):\t%f ms\n", size, size, (finalTime - initialTime));
    // Free device buffers
    hipFree(image);
    hipFree(hist);
    return 0;
}
