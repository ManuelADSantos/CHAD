#include "hip/hip_runtime.h"
// nvcc -o lab5_ex4_1 lab5_ex4_1.cu
#include <stdio.h>
#define bin 16

// Device code
__global__ void histogram(unsigned char * image, int * hist, int size)
{
    int Col = threadIdx.x + (blockIdx.x * blockDim.x);
    int Row = threadIdx.y + (blockIdx.y * blockDim.y);

    int i = Row * size + Col;
    unsigned char pixel = image[i];

    int index = (int)(pixel/bin);

    atomicAdd(&hist[index],1);
}

int main()
{
    int size = 256;
    int N = size * size;

    //inicializa��o da imagem
    unsigned char aux[N];
    int aux2[bin];
    for (int i=0; i<N; i++)
    {
        aux[i] = 255 * sin(i) * sin(i);
    }

    for (int i=0; i<bin; i++)
    {
        aux2[i] = 0;
    }

    //aloca��o de mem�ria para a GPU
    unsigned char *image = NULL;
    int *hist = NULL;

    int err1 = 0;
    int err2 = 0;
    err1 = hipMalloc(&image, sizeof(unsigned char)*N);
    err2 = hipMalloc(&hist, sizeof(int)*bin);
    if(err1 != hipSuccess || err2 != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }

    //CLOCK_PROCESS_CPUTIME_ID - Profiling the execution time of loop
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);

    err1 = hipMemcpy(image,&aux,sizeof(unsigned char)*N,hipMemcpyHostToDevice);
    err2 = hipMemcpy(hist,&aux2,sizeof(int)*bin,hipMemcpyHostToDevice);
    if(err1 != hipSuccess || err2 != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }

    // Launch device function
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid(size/16, size/16, 1);
    histogram<<<blocksPerGrid,threadsPerBlock>>>(image, hist, size);

    // Copy data from device memory to host memory
    int * host_buffer = (int *)malloc(sizeof(int)*bin);
    err1 = hipMemcpy(host_buffer,hist,sizeof(int)*bin,hipMemcpyDeviceToHost);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }

    clock_gettime(CLOCK_MONOTONIC, &end);
	double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    int contador = 0;
	for(int i=0; i<bin; i++)
	{
		printf("hist[%i]=%u\n", i, host_buffer[i]);
		contador += host_buffer[i];
	} 
	printf("Contagem total = %i\n", contador);
    printf("Cuda (%i*%i elemens):\t%f ms\n", size, size, (finalTime - initialTime));
    // Free device buffers
    hipFree(image);
    hipFree(hist);
    return 0;
}
