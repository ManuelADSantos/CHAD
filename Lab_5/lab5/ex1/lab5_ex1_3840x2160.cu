#include "hip/hip_runtime.h"
// nvcc -o lab5_ex1_3840x2160 lab5_ex1_3840x2160.cu
#include <stdio.h>
#define CHANNELS 3

// Device code
__global__ void colorToGreyScaleConvertion(unsigned char * grayImage, unsigned char * rgbImage, int width, int height)
{
    int Col = threadIdx.x + (blockIdx.x * blockDim.x);
    int Row = threadIdx.y + (blockIdx.y * blockDim.y);
    if(Col < width && Row < height)
    {
        //get 1D coordinate for the grayscale image
        int greyOffset = Row * width + Col;
        //onre can think of the RGB image having CHANNEL times columns of the gray scale image
        int rgbOffset = greyOffset * CHANNELS;
        unsigned char r = rgbImage[rgbOffset];
        //printf("r = %u\n", r);
        //red value for pixel
        unsigned char g = rgbImage[rgbOffset + 1];
        //green value for pixel
        unsigned char b = rgbImage[rgbOffset + 2];
        //blue value for pixel
        //perform the rescaling and store it 
        //we multiply by floating point constants
        grayImage[greyOffset] = (unsigned char)(0.21f * r + 0.71f * g + 0.07f * b);
    }
}

int main()
{
    int width = 3840;
    int height = 2160;
    int N = width * height;

    //inicializa��o da imagem
    unsigned char aux[N * CHANNELS];
    for (int i=0; i<N; i++)
    {
        aux[i*CHANNELS] = 255 * sin(i) * sin(i);
        aux[i*CHANNELS + 1] = 255 * sin(i) * sin(i);
        aux[i*CHANNELS + 2] = 255 * sin(i) * sin(i);
    }

    //aloca��o de mem�ria para a GPU
    unsigned char *rgbImage = NULL;
    unsigned char *grayImage = NULL;

    int err1 = 0;
    int err2 = 0;
    err1 = hipMalloc(&rgbImage, sizeof(unsigned char)*N*CHANNELS);
    err2 = hipMalloc(&grayImage, sizeof(unsigned char)*N);
    if(err1 != hipSuccess || err2 != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }

    //CLOCK_PROCESS_CPUTIME_ID - Profiling the execution time of loop
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);

    err1 = hipMemcpy(rgbImage,&aux,sizeof(unsigned char)*N*CHANNELS,hipMemcpyHostToDevice);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }

    // Launch device function
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid(width/16, height/16, 1);
    colorToGreyScaleConvertion<<<blocksPerGrid,threadsPerBlock>>>(grayImage, rgbImage, width, height);
    
    // Copy data from device memory to host memory
    unsigned char * host_buffer = (unsigned char *)malloc(sizeof(unsigned char)*N);
    err1 = hipMemcpy(host_buffer,grayImage,sizeof(unsigned char)*N,hipMemcpyDeviceToHost);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }
    clock_gettime(CLOCK_MONOTONIC, &end);
	double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    printf("host_buffer[%i][%i]=%u\n", height/2, width/2, host_buffer[((height/2) * width) + width/2]);
    printf("host_buffer[%i][%i]=%u\n", height/4, width/4, host_buffer[((height/4) * width) + width/4]);
    printf("host_buffer[%i][%i]=%u\n", 3*height/4, 3*width/4, host_buffer[((3*height/4) * width) + 3*width/4]);
    printf("Cuda (%i*%i elemens):\t%f ms\n", width, height, (finalTime - initialTime));
    // Free device buffers
    hipFree(rgbImage);
    hipFree(grayImage);
    return 0;
}
