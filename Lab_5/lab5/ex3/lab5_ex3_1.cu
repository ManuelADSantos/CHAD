#include "hip/hip_runtime.h"
// nvcc -o lab5_ex3_1 lab5_ex3_1.cu
#include <stdio.h>

// Device code
__global__ void reduce(int *g_idata, int *g_odata, int width, int height)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    // CODE
    g_odata[col * height + row] = g_idata[row * width + col];
}

int main()
{
    int width = 512;
    int height = 512;
    int N = width * height;

    //inicializa��o do vetor
    int aux[N];
    for(int i=0; i<height; i++)
	{
		for (int j=0; j<width; j++)
		{
			aux[i*width + j] = i;
		}
	} 
    int *v1 = NULL;
    int *v2 = NULL;
    int err1 = 0;
    int err2 = 0;
    err1 = hipMalloc(&v1, sizeof(int)*N);
    err2 = hipMalloc(&v2, sizeof(int)*(N));
    if(err1 != hipSuccess || err2 != hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }
    //CLOCK_PROCESS_CPUTIME_ID - Profiling the execution time of loop
	struct timespec start, end;
	clock_gettime(CLOCK_MONOTONIC, &start);

    err1 = hipMemcpy(v1,&aux,sizeof(int)*N,hipMemcpyHostToDevice);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data to device memory.\n");
    }
    // Launch device function
    dim3 threadsPerBlock(16, 16, 1);
    dim3 blocksPerGrid(width/16, height/16, 1);
    reduce<<<blocksPerGrid,threadsPerBlock>>>(v1, v2, width, height);
    
    // Copy data from device memory to host memory
    int * host_buffer = (int *)malloc(sizeof(int)*N);
    err1 = hipMemcpy(host_buffer,v2,sizeof(int)*N,hipMemcpyDeviceToHost);
    if(err1 != hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }
    clock_gettime(CLOCK_MONOTONIC, &end);
	double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
	double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    printf("host_buffer[%i][%i]=%u\n", width/2, height/4, host_buffer[((width/2) * height) + height/4]);
    printf("host_buffer[%i][%i]=%u\n", width/4, 3*height/4, host_buffer[((width/4) * height) + 3*height/4]);
    printf("host_buffer[%i][%i]=%u\n", 3*width/4, height/2, host_buffer[((3*width/4) * height) + height/2]);
    printf("Cuda (%ix%i elemens):\t%f ms\n", width, height, (finalTime - initialTime));
    // Free device buffers
    hipFree(v1);
    hipFree(v2);
    return 0;
}
