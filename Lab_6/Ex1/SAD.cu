/*===================================================================================
	SAD.C
=====================================================================================

Contains:

Written by: Oscar Ferraz
			University of Coimbra - Electrical Computer end Engineering Department
			Instituto de Telecomunicações
			email: <oscar.ferraz@co.it.pt>
			
Date: October 2023
=====================================================================================*/



//==============================================//
// I N C L U D E S								//
//==============================================//
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <string.h>
#include <time.h>
#include <assert.h>
#include <errno.h>
//#include <helper_cuda.h>
#include <hip/hip_runtime.h>  

// Defines
#define W 1920
#define H 1080
#define NUM_FRAMES 500
#define WINDOW_SIZE 21



unsigned char frame[H][W] = {0};


__global__ void GPU_AD(unsigned char * d_v_left, unsigned char * d_v_right, signed char * d_differences);

__global__ void GPU_S(unsigned int * d_result, signed char * d_differences, unsigned short d_index);


//**************************************************************************************************
//Absolute differences
__global__ void GPU_AD(unsigned char * d_v_left, unsigned char * d_v_right, signed char * d_differences){

    unsigned short x=threadIdx.x+blockIdx.x*blockDim.x;
    unsigned short y=threadIdx.y+blockIdx.y*blockDim.y;
    unsigned short z=threadIdx.z+blockIdx.z*blockDim.z;


    d_differences[(z*H*W)+(y*W)+x]=d_v_left[(y*W)+x]-d_v_right[(z*H*W)+(y*W)+x];


    if(d_differences[(z*H*W)+(y*W)+x]<0)
        d_differences[(z*H*W)+(y*W)+x]=-d_differences[(z*H*W)+(y*W)+x]; 
} 


//**************************************************************************************************
//Sum
__global__ void GPU_S(unsigned int * d_result, signed char * d_differences, unsigned short d_index){

    register unsigned int sum=0;

    for(int y=0; y<H; y++){
        for(int x=0; x<W; x++){
            sum=sum+d_differences[(threadIdx.x*H*W)+(y*W)+x];
        }
    }
    __syncthreads();
    d_result[(d_index*WINDOW_SIZE)+threadIdx.x]=sum;
    
} 



/*************************************************************************************************************/
/*																											 */
/*  M A I N   P R O G R A M																					 */		
/*																											 */
/*************************************************************************************************************/
int main(){

    hipError_t err=hipSuccess; 

    //======================================================================================================================================================================
    //kernel dimensions

    dim3 threadsPerBlock_AD(W/2,1,1);
    dim3 numBlocks_AD(2,H,WINDOW_SIZE);

    dim3 threadsPerBlock_S(WINDOW_SIZE,1,1);
    dim3 numBlocks_S(1,1,1);

    //======================================================================================================================================================================
    //size of variables


    size_t size_video=(sizeof(unsigned char)*W*H*NUM_FRAMES);
    size_t size_differences=(sizeof(signed char)*W*H*NUM_FRAMES);  
    size_t size_left=(sizeof(unsigned char)*W*H); 
    size_t size_right=(sizeof(unsigned char)*W*H*WINDOW_SIZE); 
    size_t size_result=(sizeof(unsigned int)*NUM_FRAMES*WINDOW_SIZE); 

    //==================================================================================================================LPi====================================================
    //variables declaration

    unsigned char *h_v_left=NULL;
    unsigned char *h_v_right=NULL;
    unsigned int *h_result=NULL;
    unsigned char *d_v_left=NULL;
    unsigned char *d_v_right=NULL;
    unsigned int *d_result=NULL;
    signed char *d_differences=NULL;
    unsigned int *d_index=NULL;

    //======================================================================================================================================================================
    //allocate host memory

    err=hipHostAlloc((void **)&h_v_left, size_video, hipHostMallocDefault );
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate host h_v_left(error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipHostAlloc((void **)&h_v_right, size_video, hipHostMallocDefault );
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate host h_v_right(error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipHostAlloc((void **)&h_result, size_result, hipHostMallocDefault );
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate host h_result(error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    //======================================================================================================================================================================
    //allocate device memory

    err=hipMalloc((void **)&d_v_left, size_left);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate device d_v_left (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMalloc((void **)&d_v_right, size_right);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate device d_v_right (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMalloc((void **)&d_result, size_result);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate device d_result (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMalloc((void **)&d_differences, size_differences);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate device d_differences (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipMalloc((void **)&d_index, sizeof(unsigned short));
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to allocate device d_differences (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    //======================================================================================================================================================================
    //Load Left video
    int x, y, count, index=0;
     
    FILE *pipein = popen("ffmpeg -i Left_gray.mp4 -f image2pipe -vcodec rawvideo -vframes 500 -pix_fmt gray -", "r");
     
    // Process video frames
    while(1)
    {
        // Read a frame from the input pipe into the buffer
        count = fread(frame, 1, H*W, pipein);
         
        // If we didn't get a frame of video, we're probably at the end
        if (count != H*W)break;
         
        // Process this frame
        for (y=0 ; y<H ; ++y){
            for (x=0 ; x<W ; ++x){
                h_v_left[(index*H*W)+(y*W)+x]=frame[y][x];
                
            }
        }
        index++;
    }
     
    // Flush and close input and output pipes
    fflush(pipein);
    pclose(pipein);


    //======================================================================================================================================================================
    //Load Right video
    index=0;
     
    // Open an input pipe from ffmpeg 
    FILE *pipein2 = popen("ffmpeg -i Right_gray.mp4 -f image2pipe -vcodec rawvideo -vframes 500 -pix_fmt gray -", "r");
     
    // Process video frames
    while(1)
    {
        // Read a frame from the input pipe into the buffer
        count = fread(frame, 1, H*W, pipein2);
         
        // If we didn't get a frame of video, we're probably at the end
        if (count != H*W)break;
         
        // Process this frame
        for (y=0 ; y<H ; ++y){
            for (x=0 ; x<W ; ++x){
                h_v_right[(index*H*W)+(y*W)+x]=frame[y][x];
                
            }
        }
        index++;
    }
     
    // Flush and close input and output pipes
    fflush(pipein2);
    pclose(pipein2);



    //======================================================================================================================================================================
    //Main loop
    for (int j=((WINDOW_SIZE-1)/2); j< NUM_FRAMES-((WINDOW_SIZE-1)/2); j++){

        //======================================================================================================================================================================
        //copy data to device
        err=hipMemcpy(d_v_left, h_v_left + (W*H*j), size_left, hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            fprintf(stderr, "Failed to copy the d_v_left from host to device (error code %d)!\n", hipGetLastError());
            exit(EXIT_FAILURE);
        }

        
        printf("frames=%d\n", (j - ((WINDOW_SIZE-1)/2)));
        err=hipMemcpy(d_v_right, h_v_right + (W*H*(j - ((WINDOW_SIZE-1)/2))) , size_right, hipMemcpyHostToDevice);
        if(err!=hipSuccess){
            fprintf(stderr, "Failed to copy the d_v_right from host to device (error code %d)!\n", hipGetLastError());
            exit(EXIT_FAILURE);
        }

        //======================================================================================================================================================================
        //execute the kernel
        GPU_AD<<<numBlocks_AD, threadsPerBlock_AD>>>(d_v_left, d_v_right, d_differences);
        if(err!=hipSuccess){
            fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
            exit(EXIT_FAILURE);
        }

        hipDeviceSynchronize();

        GPU_S<<<numBlocks_S, threadsPerBlock_S>>>( d_result, d_differences, j - ((WINDOW_SIZE/2)-1));
        if(err!=hipSuccess){
            fprintf(stderr, "Failed to launch the kernel (error code %d)!\n", hipGetLastError());
            exit(EXIT_FAILURE);
        }

        hipDeviceSynchronize();
        printf("frame=%d\n", j);
    }

    //======================================================================================================================================================================
    //copy the data from device to host
    err=hipMemcpy(h_result , d_result, size_result, hipMemcpyDeviceToHost);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to copy the result from device to host (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    } 

    //======================================================================================================================================================================
    //Write histogram to file
    FILE *fptr;

    fptr = fopen("file.txt","w");

    if(fptr == NULL)
    {
        printf("Error!");   
        exit(1);             
    }

    for(int y=0; y<NUM_FRAMES; y++){
        for(int x=0; x<WINDOW_SIZE; x++){
            fprintf(fptr,"%d\t",h_result[(y*WINDOW_SIZE)+x]);
        }
        fprintf(fptr,"\n");
    }

    fclose(fptr); 


    //======================================================================================================================================================================
    //free the device memory

    err=hipFree(d_v_left);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the d_v_left from the device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipFree(d_v_right);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the d_v_right from the device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipFree(d_result);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the d_result from the device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipFree(d_differences);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the d_differences from the device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipFree(d_index);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the d_differences from the device (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }


    //======================================================================================================================================================================
    //free the host memory

    err=hipHostFree(h_v_left);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the h_v_left from the host (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipHostFree(h_v_right);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the h_v_right from the host (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }

    err=hipHostFree(h_result);
    if(err!=hipSuccess){
        fprintf(stderr, "Failed to free the h_result from the host (error code %d)!\n", hipGetLastError());
        exit(EXIT_FAILURE);
    }


    return(0);
}


