// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 07/11/2023
// ============================================================================
// -> Compile
// nvcc -o Lab6_ex4_d Lab6_ex4_d.cu -lrt
// ============================================================================

#include <stdio.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

__global__ void get_pi(unsigned long long int *count_circle, unsigned long long int *count_square, int seed)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    for(int i = 0; i < 100000; i++)
    {
        float x = hiprand_uniform(&state)*2.0 - 1.0;
        float y = hiprand_uniform(&state)*2.0 - 1.0;

        if (x*x + y*y <= 1.0)
            atomicAdd(count_circle, 1);
        else
            atomicAdd(count_square, 1);
    }
}


int main(int argc, char *argv[])
{   
    unsigned long long int circle_host = 0, square_host = 0;
    unsigned long long int *circle_device, *square_device;
    int time_host = time(NULL);

    hipMalloc((void**)&circle_device, sizeof(unsigned long long int));
    hipMalloc((void**)&square_device, sizeof(unsigned long long int));

    hipMemcpy(circle_device, &circle_host, sizeof(unsigned long long int), hipMemcpyHostToDevice);
    hipMemcpy(square_device, &square_host, sizeof(unsigned long long int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = 256;

    get_pi<<<blocksPerGrid, threadsPerBlock>>>(circle_device, square_device, time_host);

    hipMemcpy(&circle_host, circle_device, sizeof(unsigned long long int), hipMemcpyDeviceToHost);
    hipMemcpy(&square_host, square_device, sizeof(unsigned long long int), hipMemcpyDeviceToHost);

    printf("Pi = %.10lf\n", (double)(4.0*circle_host/(double)(circle_host+square_host)));
    printf("Points: Circle = %llu || Square = %llu\n", circle_host, square_host);

    hipFree(circle_device);
    hipFree(square_device);

    return 0;
}