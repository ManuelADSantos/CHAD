// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 07/11/2023
// ============================================================================
// -> Compile
// nvcc -o Lab6_ex2 Lab6_ex2.cu -lrt
// ============================================================================

#include <stdio.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

#define threadsPerBlock 256
#define blocksPerGrid 1024
#define totalThreads (threadsPerBlock*blocksPerGrid)
#define triesPerThread 1000000

// ==================== KERNEL ====================
__global__ void get_pi(unsigned long long int *count_circle, unsigned long long int *count_square, int seed)
{
    // ===== Calculate global index
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    // ===== Initialize random number generator
    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    // ===== Initialize counters
    count_circle[idx] = 0;
    count_square[idx] = 0;

    // ===== Generate random numbers and count
    for(int i = 0; i < triesPerThread; i++)
    {
        float x = hiprand_uniform(&state)*2.0 - 1.0;
        float y = hiprand_uniform(&state)*2.0 - 1.0;

        if (x*x + y*y <= 1.0)
            count_circle[idx]++;
        else
            count_square[idx]++;
    }
}


// ==================== MAIN ====================
int main(int argc, char *argv[])
{   
    // ===== Variables
    // -- Host
    unsigned long long int *circle_host = NULL, *square_host = NULL;
    // -- Device
    unsigned long long int *circle_device = NULL, *square_device = NULL;
    // -- Random seed
    int rand_seed = time(NULL);

    // ===== Alocate memory in host
    circle_host = (unsigned long long int*)malloc(sizeof(unsigned long long int)*totalThreads);
    square_host = (unsigned long long int*)malloc(sizeof(unsigned long long int)*totalThreads);

    // ===== Alocate memory in device
    hipMalloc((void**)&circle_device, sizeof(unsigned long long int)*totalThreads);
    hipMalloc((void**)&square_device, sizeof(unsigned long long int)*totalThreads);

    // ===== Print info
    printf("Threads per block: %d || Blocks per grid: %d\n", threadsPerBlock, blocksPerGrid);

    // ===== Call kernel
    get_pi<<<blocksPerGrid, threadsPerBlock>>>(circle_device, square_device, rand_seed);

    // ===== Copy memory from device to host
    hipMemcpy(circle_host, circle_device, sizeof(unsigned long long int)*totalThreads, hipMemcpyDeviceToHost);
    hipMemcpy(square_host, square_device, sizeof(unsigned long long int)*totalThreads, hipMemcpyDeviceToHost);

    // ===== Sum all results
    unsigned long long int circle_sum = 0, square_sum = 0;
    for(int i = 0; i < totalThreads; i++)
    {
        circle_sum += circle_host[i];
        square_sum += square_host[i];
    }

    // ===== Print results and validate results (pi = 3.141 592 653 58979323846)
    printf("--> Pi = %.10lf\n", (double)(4.0*circle_sum/(double)(circle_sum + square_sum)));
    printf("Points: Circle = %llu\n        Square = %llu\n", circle_sum, square_sum);

    // ===== Free memory
    // -- Host
    free(circle_host);
    free(square_host);
    // -- Device
    hipFree(circle_device);
    hipFree(square_device);

    return 0;
}