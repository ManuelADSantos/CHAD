// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 07/11/2023
// ============================================================================
// -> Compile
// nvcc -o Lab6_ex2 Lab6_ex2.cu
// ============================================================================


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__global__ void get_pi(int *count_circle, int *count_square)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState state;
    hiprand_init(1234, idx, 0, &state);

    float x = hiprand_uniform(&state)*2.0 - 1.0;
    float y = hiprand_uniform(&state)*2.0 - 1.0;

    if (x*x + y*y <= 1.0)
        atomicAdd(count_circle, 1);
    else
        atomicAdd(count_square, 1);
}


int main(int argc, char *argv[])
{   
    int circle_host = 0, square_host = 0;
    int *circle_device, *square_device;

    hipMalloc((void**)&circle_device, sizeof(int));
    hipMalloc((void**)&square_device, sizeof(int));

    hipMemcpy(circle_device, &circle_host, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(square_device, &square_host, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = 2000;

    get_pi<<<blocksPerGrid, threadsPerBlock>>>(circle_device, square_device);

    hipMemcpy(&circle_host, circle_device, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&square_host, square_device, sizeof(int), hipMemcpyDeviceToHost);

    printf("Pi = %f\n", 4.0*circle_host/(circle_host+square_host));

    return 0;
}