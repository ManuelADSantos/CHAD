// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 07/11/2023
// ============================================================================
// -> Compile
// nvcc -o Lab6_ex2 Lab6_ex2.cu -lrt
// ============================================================================

#include <stdio.h>
#include <time.h>
#include <hiprand/hiprand_kernel.h>
#include <hip/hip_runtime.h>

__global__ void get_pi(int *count_circle, int *count_square, int seed)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState state;
    hiprand_init(seed, idx, 0, &state);

    for(int i = 0; i < 1000; i++)
    {
        float x = hiprand_uniform(&state)*2.0 - 1.0;
        float y = hiprand_uniform(&state)*2.0 - 1.0;

        if (x*x + y*y <= 1.0)
            atomicAdd(count_circle, 1);
        else
            atomicAdd(count_square, 1);
    }
}


int main(int argc, char *argv[])
{   
    int circle_host = 0, square_host = 0;
    int *circle_device, *square_device;
    int time_host = time(NULL);

    hipMalloc((void**)&circle_device, sizeof(int));
    hipMalloc((void**)&square_device, sizeof(int));

    hipMemcpy(circle_device, &circle_host, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(square_device, &square_host, sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocksPerGrid = 256;

    get_pi<<<blocksPerGrid, threadsPerBlock>>>(circle_device, square_device, time_host);

    hipMemcpy(&circle_host, circle_device, sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(&square_host, square_device, sizeof(int), hipMemcpyDeviceToHost);

    printf("Pi = %f\n", 4.0*circle_host/(circle_host+square_host));

    return 0;
}