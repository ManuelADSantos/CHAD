// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 07/11/2023
// ============================================================================
// -> Compile
// nvcc -o test_random_number test_random_number.cu
// ============================================================================


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>

__global__ void get_pi(float *results)
{
    int idx = threadIdx.x + blockIdx.x * blockDim.x;

    hiprandState state;
    hiprand_init(1234, idx, 0, &state);
    results[idx] = hiprand_uniform(&state);    
}


int main(int argc, char *argv[])
{   
    int N = 256;

    float *result_host, *result_device;

    result_host = (float*)malloc(N*sizeof(float));
    hipMalloc((void**)&result_device, N*sizeof(float));

    get_pi<<<1, N>>>(result_device);

    hipMemcpy(result_host, result_device, N*sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++)
    {
        printf("[%d]: %g\n", i, result_host[i]);
    }

    return 0;
}