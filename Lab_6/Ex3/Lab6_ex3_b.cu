#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 07/11/2023
// ============================================================================
// -> Compile
// nvcc -o Lab6_ex3_b Lab6_ex3_b.cu -lrt
// ============================================================================

// ===== Images Library
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image.h"
#include "lib/stb_image_write.h"
#include <stdio.h>
#include <stdlib.h>

// ===== Kernel Properties
#define BLUR_SIZE 3

// ======================================== KERNEL ========================================
// ======================================== KERNEL ========================================
__global__ void blurKernel(unsigned char* in, unsigned char* out, int width, int height, int num_channel) 
{   
    // ===== Pixel Variables
    int pixSum, numPixels;

    // ===== Global Pixel Position
    int col_global = blockIdx.x * blockDim.x + threadIdx.x;
    int row_global = blockIdx.y * blockDim.y + threadIdx.y;

    // ===== Check if pixel is inside image
    if(col_global > -1 && col_global < width && row_global > -1 && row_global < height ) 
    {
        // ===== Iterate over all color channels
        for(int channel = 0; channel < num_channel; channel++)
        {
            // ===== Shared Memory
            __shared__ unsigned char tile[BLUR_SIZE*BLUR_SIZE];

            // ===== Local Pixel Position
            int col_local = 0, row_local = 0;

            for(int i = -BLUR_SIZE/2; i <= BLUR_SIZE/2; i++)
            {
                for(int j = -BLUR_SIZE/2; j <= BLUR_SIZE/2; j++)
                {
                    // ===== Check if pixel is inside image
                    if((row_global + i) < 0 || (row_global + i) >= height || (col_global + j) < 0 || (col_global + j) >= width)
                        tile[row_local * BLUR_SIZE + col_local] = 0;
                    else
                        tile[row_local * BLUR_SIZE + col_local] = in[((row_global + i) * width  + (col_global + j)) * num_channel + channel];
                    
                    __syncthreads();
                    col_local++;
                }
                row_local++;
            }

            // ===== Calculate Pixel Sum
            pixSum = 0;
            numPixels = 0;
            for(int i = 0; i < BLUR_SIZE; i++)
            {
                for(int j = 0; j < BLUR_SIZE; j++)
                {
                    pixSum += tile[i * BLUR_SIZE + j];
                    __syncthreads();
                    numPixels++;
                }
            }

            // ===== Calculate Pixel Average
            out[row_global * width * num_channel + col_global * num_channel + channel] = (unsigned char)(pixSum / numPixels);
        }
    }
}


// ======================================== MAIN ========================================
int main(int argc, char *argv[])
{
    // ===== Get correct image
    int img_id = atoi(argv[1]);
    char img_name[50];
    sprintf(img_name, "images/in/image%d.jpg", img_id);

    // ===== Image Properties
    int width, height, n;

    // ===== Load Original Image
    unsigned char *image = stbi_load(img_name,&width,&height,&n,0);
    
    // ===== Allocate Memory for Blurred Image
    unsigned char *output = (unsigned char*)malloc(width * height * n *sizeof(unsigned char));
    
    // ===== Allocate Device Memory
    unsigned char* Dev_Input_Image = NULL;
    unsigned char* Dev_Output_Image = NULL;
    hipMalloc((void**)&Dev_Input_Image, sizeof(unsigned char)* height * width * n);
    hipMalloc((void**)&Dev_Output_Image, sizeof(unsigned char)* height * width * n);
    
    // ===== Copy Host Image to Device Image
    hipMemcpy(Dev_Input_Image, image, sizeof(unsigned char) * height * width * n, hipMemcpyHostToDevice);
    
    // ===== Kernel Dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // ===== Start Time
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    // ===== Kernel Call
    blurKernel <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height, n);
    
    // ===== End Time
    clock_gettime(CLOCK_MONOTONIC, &end);

    // ===== Copy Device Image to Host Image
    hipMemcpy(image, Dev_Output_Image, sizeof(unsigned char) * height * width * n, hipMemcpyDeviceToHost);
    
    // ===== Save Blurred Image
    sprintf(img_name, "images/out/image%d_shared.jpg", img_id);
    stbi_write_jpg(img_name, width, height, n, image, width * n);
    
    // ===== Print Time Results
    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
    double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    printf("Time of execution: %f ms\n", (finalTime - initialTime));
    
    // ===== Free Device Memory
    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);
    
    return 0;
}