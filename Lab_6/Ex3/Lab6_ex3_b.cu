#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 07/11/2023
// ============================================================================
// -> Compile
// nvcc -o Lab6_ex3_b Lab6_ex3_b.cu -lrt
// ============================================================================

#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image.h"
#include "lib/stb_image_write.h"

#define BLUR_SIZE 16
#define R 0
#define G 1
#define B 2
#define TILE_DIM 16
#define BLOCK_SIZE 16

__global__ void blurKernel_shared(unsigned char* in, unsigned char* out, int width, int height, int num_channel, int channel)
{
    __shared__ unsigned char tile[BLOCK_SIZE*BLOCK_SIZE];
    int col = blockIdx.x * TILE_DIM + threadIdx.x;
    int row = blockIdx.y * TILE_DIM + threadIdx.y;
    int pixVal;
    int pixels;

    if(col > -1 && col < width && row > -1 && row < height )
    {
        pixVal = 0;
        pixels = 0;
        tile[row*width*num_channel + col*num_channel] = in[row*width*num_channel + col*num_channel];
        __syncthreads();
        for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow)
        {
            for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol)
            {
                int curRow = row + blurRow;
                int curCol = col + blurCol;
                if(curRow > -1 && curRow < height && curCol > -1 && curCol < width)
                {
                    pixVal += tile[curRow * width * num_channel + curCol * num_channel + channel];
                    pixels++;
                    __syncthreads();
                }
            }
        }
        out[row * width * num_channel + col * num_channel + channel] = (unsigned char)(pixVal/pixels);
    }
}

// ==================== MAIN ==================== 
int main(int argc, char *argv[])
{
    // ===== Get correct image
    int img_id = atoi(argv[1]);
    char img_name[50];
    sprintf(img_name, "images/in/image%d.jpg", img_id);

    int width, height, n;
    unsigned char *image = stbi_load(img_name,&width,&height,&n,0);

    // printf("Image width: %dpx, height: %dpx, channels: %d\n", width, height, n);
    // return 0;

    unsigned char *output = (unsigned char*)malloc(width * height * n *sizeof(unsigned char));
    unsigned char* Dev_Input_Image = NULL;
    unsigned char* Dev_Output_Image = NULL;
    
    hipMalloc((void**)&Dev_Input_Image, sizeof(unsigned char)* height * width * n);
    hipMalloc((void**)&Dev_Output_Image, sizeof(unsigned char)* height * width * n);
    //kernel call
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize(width/blockSize.x+1, height/blockSize.y+1);
    
    struct timespec start, end;
    
    //b)
    hipMalloc((void**)&Dev_Input_Image, sizeof(unsigned char)* height * width * n);
    hipMalloc((void**)&Dev_Output_Image, sizeof(unsigned char)* height * width * n);
    clock_gettime(CLOCK_MONOTONIC, &start);
    hipMemcpy(Dev_Input_Image, image, sizeof(unsigned char) * height * width * n, hipMemcpyHostToDevice);

    blurKernel_shared <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height, n, 0);
    blurKernel_shared <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height,n,1);
    blurKernel_shared <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height,n,2);
    hipDeviceSynchronize(); // we need this so the kernel is guaranteed to finish (and the output from the kernel will find a waiting standard output queue), before the application is allowed to exit
    hipMemcpy(image, Dev_Output_Image, sizeof(unsigned char) * height * width * n, hipMemcpyDeviceToHost);
    clock_gettime(CLOCK_MONOTONIC, &end);
    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);

    sprintf(img_name, "images/out/image%d_shared.jpg", img_id);
    stbi_write_jpg(img_name, width, height, n, image, width * n);

    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
    double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    printf("Time of execution: %f ms\n", (finalTime - initialTime));
    return 0;
}