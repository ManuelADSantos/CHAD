#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 07/11/2023
// ============================================================================
// -> Compile
// nvcc -o Lab6_ex3_b Lab6_ex3_b.cu -lrt
// ============================================================================

// ===== Images Library
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image.h"
#include "lib/stb_image_write.h"
#include <stdio.h>
#include <stdlib.h>

// ===== Kernel Properties
#define BLUR_SIZE 5
#define TILE_SIZE 16

// ======================================== KERNEL ========================================
__global__ void blurKernel(unsigned char* in, unsigned char* out, int width, int height, int num_channel) 
{   
    // ===== Global Pixel Position
    int idx_Global = blockIdx.x * blockDim.x + threadIdx.x;

    // ===== Shared Memory
    __shared__ unsigned char shared_Image[TILE_SIZE*TILE_SIZE][BLUR_SIZE][BLUR_SIZE][3];

    // ===== Work on each color channel
    for (int channel = 0; channel < num_channel; channel++)
    {   
        // ===== Shared Memory Position
        int local_row = 0, local_col = 0, count = 0;

        // ===== Horizontal offset from Global Pixel
        for (int i = -BLUR_SIZE / 2; i <= BLUR_SIZE / 2; i++)
        {
            // ===== Vertical offset from Global Pixel
            for (int j = -BLUR_SIZE / 2; j <= BLUR_SIZE / 2; j++)
            {
                // ===== Out of Bounds of Image
                if (idx_Global + i*width + j < 0 || idx_Global + i*width + j >= width*height)
                {
                    shared_Image[threadIdx.x][local_row][local_col][channel] = 0;  
                    if (idx_Global == 0)
                }
                // ===== In Bounds of Image
                else
                    shared_Image[threadIdx.x][local_row][local_col][channel] = (unsigned char)in[(idx_Global + i*width + j) * num_channel + channel];
                    count++;
                
                // ===== Update Local Position (in kernel)
                local_col++;
            }
            // ===== Update Local Position (in kernel)
            local_row++;
            local_col = 0;
        }
        __syncthreads();
            
        // ===== Blur Pixel
        int sum = 0;
        for (int i = 0; i < BLUR_SIZE; i++)
        {
            for (int j = 0; j < BLUR_SIZE; j++)
            {
                sum += shared_Image[threadIdx.x][i][j][channel];
            }
        }
        __syncthreads();

        // ===== Save Blurred Pixel
        out[idx_Global * num_channel + channel] = (unsigned char)(sum / count);
        __syncthreads();
    }
}



// ======================================== MAIN ========================================
int main(int argc, char *argv[])
{
    // ===== Get correct image
    int img_id = atoi(argv[1]);
    char img_name[50];
    sprintf(img_name, "images/in/image%d.jpg", img_id);

    // ===== Image Properties
    int width, height, n;

    // ===== Load Original Image
    unsigned char *image = stbi_load(img_name,&width,&height,&n,0);

    // ===== Allocate Memory for Blurred Image
    unsigned char *output = (unsigned char*)malloc(width * height * n *sizeof(unsigned char));

    // ===== Allocate Device Memory
    unsigned char* Dev_Input_Image = NULL;
    unsigned char* Dev_Output_Image = NULL;
    hipMalloc((void**)&Dev_Input_Image, sizeof(unsigned char)* height * width * n);
    hipMalloc((void**)&Dev_Output_Image, sizeof(unsigned char)* height * width * n);
    
    // ===== Copy Host Image to Device Image
    hipMemcpy(Dev_Input_Image, image, sizeof(unsigned char) * height * width * n, hipMemcpyHostToDevice);
    
    // ===== Kernel Dimensions
    int threadsPerBlock = TILE_SIZE * TILE_SIZE;
    int BlocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    // ===== Start Time
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    // ===== Kernel Call
    blurKernel <<<BlocksPerGrid, threadsPerBlock>>>(Dev_Input_Image, Dev_Output_Image, width, height, n);
    
    // ===== End Time
    clock_gettime(CLOCK_MONOTONIC, &end);

    // ===== Copy Device Image to Host Image
    hipMemcpy(image, Dev_Output_Image, sizeof(unsigned char) * height * width * n, hipMemcpyDeviceToHost);
    
    // ===== Save Blurred Image
    sprintf(img_name, "images/out/image%d_shared.jpg", img_id);
    stbi_write_jpg(img_name, width, height, n, image, width * n);
    
    // ===== Print Time Results
    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
    double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    printf("Time of execution: %f ms\n", (finalTime - initialTime));
    
    // ===== Free Device Memory
    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);
    
    return 0;
}