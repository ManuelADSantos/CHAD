#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 07/11/2023
// ============================================================================
// -> Compile
// nvcc -o Lab6_ex3_b Lab6_ex3_b.cu -lrt
// ============================================================================

// ===== Images Library
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image.h"
#include "lib/stb_image_write.h"
#include <stdio.h>
#include <stdlib.h>

// ===== Kernel Properties
#define BLUR_SIZE 5
#define TILE_SIZE 16

// ======================================== KERNEL ========================================
__global__ void blurKernel(unsigned char* in, unsigned char* out, int width, int height, int num_channel) 
{   
    // ===== Global Pixel Position
    int idx_Global = blockIdx.x * blockDim.x + threadIdx.x;

    // ===== Shared Memory
    __shared__ unsigned char shared_Image[TILE_SIZE*TILE_SIZE][BLUR_SIZE][BLUR_SIZE];

    // if (idx_Global == 0)
    //     printf("%i\n", TILE_SIZE*TILE_SIZE);

    // if (blockIdx.x == 10)
    //     printf("idx_Global: %5i || threadIdx.x = %3i || blockIdx.x = %4i\n", idx_Global, threadIdx.x, blockIdx.x);

    // ===== Work on each color channel
    for (int channel = 0; channel < num_channel; channel++)
    {   
        // ===== Shared Memory Position
        int local_row = 0, local_col = 0;

        for (int i = -BLUR_SIZE / 2; i <= BLUR_SIZE / 2; i++)
        {
            for (int j = -BLUR_SIZE / 2; j <= BLUR_SIZE / 2; j++)
            {
                if (idx_Global + i*width + j < 0 || idx_Global + i*width + j >= width*height)
                    shared_Image[threadIdx.x][local_row][local_col] = 0;
                else
                    shared_Image[threadIdx.x][local_row][local_col] = in[idx_Global + i*width + j];
                      
                local_col++;
            }
            local_row++;
            local_col = 0;
        }
        __syncthreads();

        // ===== Blur Pixel
        out[idx_Global] = shared_Image[threadIdx.x][2][2];
        __syncthreads();
    }

}



// ======================================== MAIN ========================================
int main(int argc, char *argv[])
{
    // ===== Get correct image
    int img_id = atoi(argv[1]);
    char img_name[50];
    sprintf(img_name, "images/in/image%d.jpg", img_id);

    // ===== Image Properties
    int width, height, n;

    // ===== Load Original Image
    unsigned char *image = stbi_load(img_name,&width,&height,&n,0);

    // ===== Allocate Memory for Blurred Image
    unsigned char *output = (unsigned char*)malloc(width * height * n *sizeof(unsigned char));
    
    // ===== Allocate Device Memory
    unsigned char* Dev_Input_Image = NULL;
    unsigned char* Dev_Output_Image = NULL;
    hipMalloc((void**)&Dev_Input_Image, sizeof(unsigned char)* height * width * n);
    hipMalloc((void**)&Dev_Output_Image, sizeof(unsigned char)* height * width * n);
    
    // ===== Copy Host Image to Device Image
    hipMemcpy(Dev_Input_Image, image, sizeof(unsigned char) * height * width * n, hipMemcpyHostToDevice);
    
    // ===== Kernel Dimensions
    int threadsPerBlock = TILE_SIZE * TILE_SIZE;
    int BlocksPerGrid = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    // printf("Threads per block: %i\n", threadsPerBlock);
    // printf("Blocks per grid: %i\n", BlocksPerGrid);

    // printf("width: %d || height: %d || n: %d\n", width, height, n);

    // ===== Start Time
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    // ===== Kernel Call
    blurKernel <<<BlocksPerGrid, threadsPerBlock>>>(Dev_Input_Image, Dev_Output_Image, width, height, n);
    
    // ===== End Time
    clock_gettime(CLOCK_MONOTONIC, &end);

    // ===== Copy Device Image to Host Image
    hipMemcpy(image, Dev_Output_Image, sizeof(unsigned char) * height * width * n, hipMemcpyDeviceToHost);
    
    // ===== Save Blurred Image
    sprintf(img_name, "images/out/image%d_shared.jpg", img_id);
    stbi_write_jpg(img_name, width, height, n, image, width * n);
    
    // ===== Print Time Results
    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
    double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    printf("Time of execution: %f ms\n", (finalTime - initialTime));
    
    // ===== Free Device Memory
    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);
    
    return 0;
}