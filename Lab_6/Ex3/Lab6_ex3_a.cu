#include "hip/hip_runtime.h"
// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 07/11/2023
// ============================================================================
// -> Compile
// nvcc -o Lab6_ex3_a Lab6_ex3_a.cu -lrt
// ============================================================================

// ===== Images Library
#define STB_IMAGE_IMPLEMENTATION
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "lib/stb_image.h"
#include "lib/stb_image_write.h"
#include <stdio.h>
#include <stdlib.h>

// ===== Kernel Properties
#define BLUR_SIZE 5

// ======================================== KERNEL ========================================
__global__ void blurKernel(unsigned char* in, unsigned char* out, int width, int height, int num_channel) 
{   
    // ===== Pixel Variables
    int pixSum, numPixels;

    // ===== Global Pixel Position
    int col_global = blockIdx.x * blockDim.x + threadIdx.x;
    int row_global = blockIdx.y * blockDim.y + threadIdx.y;

    // ===== Check if pixel is inside image
    if(col_global > -1 && col_global < width && row_global > -1 && row_global < height ) 
    {
        // ===== Iterate over all channels
        for(int channel = 0; channel < num_channel; channel++)
        {
            // ===== Initialize Pixel Variables
            pixSum = 0;
            numPixels = 0;
            
            // ===== Iterate over row_global
            for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE + 1; ++blurRow) 
            {
                // ===== Iterate over column
                for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE + 1; ++blurCol) 
                {
                    // ===== Current Pixel Position
                    int curRow = row_global + blurRow;
                    int curCol = col_global + blurCol;

                    // ===== Check if pixel is inside filter kernel
                    if(curRow > -1 && curRow < height && curCol > -1 && curCol < width)
                    {
                        // ===== Add Pixel Value
                        pixSum += in[curRow * width * num_channel + curCol * num_channel + channel];
                        numPixels++;
                    }
                }
            }

            // ===== Save Pixel Value
            out[row_global * width * num_channel + col_global * num_channel + channel] = (unsigned char)(pixSum/numPixels);
        }
    }
}


// ======================================== MAIN ========================================
int main(int argc, char *argv[])
{
    // ===== Get correct image
    int img_id = atoi(argv[1]);
    char img_name[50];
    sprintf(img_name, "images/in/image%d.jpg", img_id);

    // ===== Image Properties
    int width, height, n;

    // ===== Load Original Image
    unsigned char *image = stbi_load(img_name,&width,&height,&n,0);
    
    // ===== Allocate Memory for Blurred Image
    unsigned char *output = (unsigned char*)malloc(width * height * n *sizeof(unsigned char));
    
    // ===== Allocate Device Memory
    unsigned char* Dev_Input_Image = NULL;
    unsigned char* Dev_Output_Image = NULL;
    hipMalloc((void**)&Dev_Input_Image, sizeof(unsigned char)* height * width * n);
    hipMalloc((void**)&Dev_Output_Image, sizeof(unsigned char)* height * width * n);
    
    // ===== Copy Host Image to Device Image
    hipMemcpy(Dev_Input_Image, image, sizeof(unsigned char) * height * width * n, hipMemcpyHostToDevice);
    
    // ===== Kernel Dimensions
    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);

    // ===== Start Time
    struct timespec start, end;
    clock_gettime(CLOCK_MONOTONIC, &start);
    
    // ===== Kernel Call
    blurKernel <<<gridSize, blockSize>>>(Dev_Input_Image, Dev_Output_Image, width, height, n);
    
    // ===== End Time
    clock_gettime(CLOCK_MONOTONIC, &end);

    // ===== Copy Device Image to Host Image
    hipMemcpy(image, Dev_Output_Image, sizeof(unsigned char) * height * width * n, hipMemcpyDeviceToHost);
    
    // ===== Save Blurred Image
    sprintf(img_name, "images/out/image%d_basic.jpg", img_id);
    stbi_write_jpg(img_name, width, height, n, image, width * n);
    
    // ===== Print Time Results
    double initialTime=(start.tv_sec*1e3)+(start.tv_nsec*1e-6);
    double finalTime=(end.tv_sec*1e3)+(end.tv_nsec*1e-6);
    printf("Time of execution: %f ms\n", (finalTime - initialTime));
    
    // ===== Free Device Memory
    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);
    
    return 0;
}