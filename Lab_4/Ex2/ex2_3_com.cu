// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 17/10/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_3_com ex2_3_com.cu
// -> Run
// ./ex2_3_com

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

// ===== Device code =====
__global__ void add_CPU(int *a, int *b, int* c)
{
    // ===== Thread identifier (1 dimensional problem)
    int index = threadIdx.x + blockIdx.x * blockDim.x;

    // ===== Kernel Code
    c[index] = a[index] + b[index];
}

// Function to compress the vector
void compress_by_4(int *original, int *compressed, int N)
{
    int regist = 0;
    for(int i = 0; i < N; i++)
    {
        char aux = (char)original[i]; 
        regist = (int)aux | regist;
        if (i % 4 == 3)
        {  
            compressed[i/4] = regist;
            regist = 0;
        }
        regist = regist << 8;
    }
}

// Function to decompress the vector
void decompress_by_4(int *compressed, int *decompressed, int N)
{    
    int idx = 0;
    for(int i = 0; i < N; i++)
    {
        decompressed[i] = (compressed[idx] >> (24-8*(i%4))) & 0xFF;
        if (i % 4 == 3)
            idx++;
    }
}

// ===== Host code =====
int main()
{
    int N = 32768;
    int N_short = N/4;
    int err = 0;

    int v1[N], v2[N], v3[N];
    int *a, *b, *c = NULL;

    // ===== Initialize host buffers
    for (int i = 0; i < N; i++)
    {
        v1[i] = rand() % 64;
        v2[i] = rand() % 64;
    }

    // ===== Initialize timer variables
	struct timespec start, end;
	// ===== Get initial time
	clock_gettime(CLOCK_MONOTONIC, &start);

    // ==== Allocate vectors to sum in device memory  
    err = hipMalloc(&a, sizeof(int)*N_short);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector A failed in device\n");
        return -1;
    }

    err = hipMalloc(&b, sizeof(int)*N_short);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector B failed in device\n");
        return -1;
    }

    // ===== Compress the vectors
    int v1_short[N/4], v2_short[N/4];
    compress_by_4(v1, v1_short, N);
    compress_by_4(v2, v2_short, N);

    // ===== Copy data from host memory to device memory
    err = hipMemcpy(a, v1_short, sizeof(int)*N_short, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        perror("Error transfering vector A from host memory.\n");
        return -1;
    }
    err = hipMemcpy(b, v2_short, sizeof(int)*N_short, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        perror("Error transfering vector B from host memory.\n");
        return -1;
    }
    
    // ===== Allocate results vector in device memory
    err = hipMalloc(&c, sizeof(int)*N_short);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector C failed in device\n");
        return -1;
    }

    // ===== Launch device function
    int threadsPerBlock = 256;
    int blocksPerGrid = (N/4)/256;
    add_CPU<<<blocksPerGrid,threadsPerBlock>>>(a, b, c);
    hipDeviceSynchronize();

    // ===== Copy data from device memory to host memory
    int v3_short[N];
    err = hipMemcpy(v3_short, c, sizeof(int)*N_short, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        perror("Error transfering vector C from device memory.\n");
        return -1;
    }

    // ===== Decompress the vector
    decompress_by_4(v3_short, v3, N);
    
    // ===== Free device buffers
    hipFree(a);
    hipFree(b);
    hipFree(c);

    // ===== Get final time
	clock_gettime(CLOCK_MONOTONIC, &end);

    // ===== Run validation
    printf("-> Validating results for N = %d...\n", N);
    for(int i = 0; i < 5; i++)
    {
        printf("v1[%i]=%i + ", i, v1[i]);
        printf("v2[%i]=%i = ", i, v2[i]);
        printf("v3[%i]=%d\n", i, v3[i]);
    }

    // ===== Calculate the elapsed time
	double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
	double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
	printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    // ===== Done
    printf("-> Done :)\n");

    return 0;
}
