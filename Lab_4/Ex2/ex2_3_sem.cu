// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 17/10/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_3_sem ex2_3_sem.cu
// -> Run
// ./ex2_3_sem


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

// ===== Device code =====
__global__ void add_CPU(int *a, int *b, int* c)
{
    // ===== Thread identifier (1 dimensional problem)
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // ===== Kernel Code
    c[index] = a[index] + b[index];
}

// ===== Host code =====
int main()
{
    int N = 32768;
    int err = 0;

    int v1[N], v2[N], v3[N];
    int *a, *b, *c = NULL;

    // ===== Initialize host buffers
    for (int i = 0; i < N; i++)
    {
        v1[i] = rand() % 64;
        v2[i] = rand() % 64;
    }

    // ===== Initialize timer variables
	struct timespec start, end;
	// ===== Get initial time
	clock_gettime(CLOCK_MONOTONIC, &start);

    // ==== Allocate vectors to sum in device memory  
    err = hipMalloc(&a, sizeof(int)*N);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector A failed in device\n");
        return -1;
    }

    err = hipMalloc(&b, sizeof(int)*N);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector B failed in device\n");
        return -1;
    }

    // ===== Copy data from host memory to device memory
    err = hipMemcpy(a, v1, sizeof(int)*N, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        perror("Error transfering vector A from host memory to device.\n");
        return -1;
    }
    err = hipMemcpy(b, v2, sizeof(int)*N, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        perror("Error transfering vector B from host memory to device.\n");
        return -1;
    }
    
    // ===== Allocate results vector in device memory
    err = hipMalloc(&c, sizeof(int)*N);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector C failed in device\n");
        return -1;
    }

    // ===== Launch device function
    int threadsPerBlock = 256;
    int blocksPerGrid = (N/4)/256;
    add_CPU<<<blocksPerGrid,threadsPerBlock>>>(a, b, c);
    hipDeviceSynchronize();

    // ===== Copy data from device memory to host memory
    err = hipMemcpy(v3, c, sizeof(int)*N, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        perror("Error transfering vector C from device memory.\n");
        return -1;
    }
    
    // ===== Free device buffers
    hipFree(a);
    hipFree(b);
    hipFree(c);

    // ===== Get final time
	clock_gettime(CLOCK_MONOTONIC, &end);
    
    // ===== Run validation
    printf("-> Validating results for N = %d...\n", N);
    for(int i = 0; i < 5; i++)
    {
        printf("v1[%i]=%i + ", i, v1[i]);
        printf("v2[%i]=%i = ", i, v2[i]);
        printf("v3[%i]=%i\n", i, v3[i]);
    }

	// ===== Calculate the elapsed time
	double initialTime = (start.tv_sec * 1e3) + (start.tv_nsec * 1e-6);
	double finalTime = (end.tv_sec * 1e3) + (end.tv_nsec * 1e-6);
	printf("-> Execution Time:\t%f ms\n", (finalTime - initialTime));

    // ===== Done
    printf("-> Done :)\n");

    return 0;
}
