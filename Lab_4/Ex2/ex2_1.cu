// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 17/10/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_1 ex2_1.cu
// -> Run
// ./ex2_1


#include <hip/hip_runtime.h>
#include <stdio.h>

// ===== Device code =====
__global__ void add_CPU(int *a, int *b, int* c)
{
    // ===== Thread identifier (1 dimensional problem)
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // ===== Kernel Code
    c[index] = a[index] + b[index];
}

// ===== Host code =====
int main()
{
    int N = 1024;
    int err = 0;

    int *v1, *v2, *v3 = NULL;
    int *a, *b, *c = NULL;

    // ==== Allocate vectors to sum in device memory 
    // v1 (host) = a (device)
    v1 = (int *)malloc(sizeof(int)*N);
    if (v1 == NULL) {
        perror("Memory allocation for vector V1 failed in host\n");
        return -1;
    }
    err = hipMalloc(&a, sizeof(int)*N);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector A failed in device\n");
        return -1;
    }
    // v2 (host) = b (device)
    v2 = (int *)malloc(sizeof(int)*N);
    if (v2 == NULL) {
        perror("Memory allocation for vector V2 failed in host\n");
        return -1;
    }
    err = hipMalloc(&b, sizeof(int)*N);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector B failed in device\n");
        return -1;
    }

    // ===== Initialize host buffers
    for (int i = 0; i < N; i++)
    {
        v1[i] = i;
        v2[i] = i;
    }

    // ===== Copy vectors to sum from host memory to device memory
    err = hipMemcpy(a, v1, sizeof(int)*N, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        perror("Error transfering vector A from host memory.\n");
        return -1;
    }
    err = hipMemcpy(b, v2, sizeof(int)*N, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        perror("Error transfering vector B from host memory.\n");
        return -1;
    }
    
    // ==== Allocate result vector in host memory
    v3 = (int *)malloc(sizeof(int)*N);
    if (v3 == NULL) {
        perror("Memory allocation for vector V3 failed in host\n");
        return -1;
    }
    // ==== Allocate result vector in device memory
    err = hipMalloc(&c, sizeof(int)*N);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector C failed in device\n");
        return -1;
    }

    // ===== Launch device function
    int threadsPerBlock = 256;
    int blocksPerGrid = N/256;
    add_CPU<<<blocksPerGrid,threadsPerBlock>>>(a, b, c);
    hipDeviceSynchronize();

    // ===== Copy data from device memory to host memory
    err = hipMemcpy(v3, c, sizeof(int)*N, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        perror("Error transfering vector C from device memory.\n");
        return -1;
    }
    
    // ===== Run validation
    printf("-> Validating results (should be double the index)...\n");
    for(int i = 0; i < 5; i++)
    {
        printf("host_buffer[%i]=%i\n", i, v3[i]);
    }

    // ===== Free host buffers
    free(v1);
    free(v2);
    free(v3);
    // ===== Free device buffers
    hipFree(a);
    hipFree(b);
    hipFree(c);

    // ===== Done
    printf("-> Done :)\n");

    return 0;
}
