// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 17/10/2023
// ============================================================================
// -> Compile
// nvcc -o ex2_2 ex2_2.cu
// -> Run
// ./ex2_2


#include <hip/hip_runtime.h>
#include <stdio.h>

// ===== Device code =====
__global__ void add_CPU(int *a, int *b, int* c)
{
    // ===== Thread identifier (1 dimensional problem)
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    
    // ===== Kernel Code
    // Initialize vectors a and b with index
    a[index] = index;
    b[index] = index;
    // Sum vectors a and b and store result in vector c
    c[index] = a[index] + b[index];
}

// ===== Host code =====
int main()
{
    int N = 1024;
    int err = 0;

    int *v3 = NULL;
    int *a, *b, *c = NULL;

    // ==== Allocate vectors to sum in device memory  
    err = hipMalloc(&a, sizeof(int)*N);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector A failed in device\n");
        return -1;
    }
    err = hipMalloc(&b, sizeof(int)*N);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector B failed in device\n");
        return -1;
    }

    
    // ==== Allocate result vector in host memory 
    v3 = (int *)malloc(sizeof(int)*N);
    if (v3 == NULL) {
        perror("Memory allocation for vector V3 failed in host\n");
        return -1;
    }
    // ==== Allocate result vector in device memory
    err = hipMalloc(&c, sizeof(int)*N);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector C failed in device\n");
        return -1;
    }

    // ===== Launch device function
    int threadsPerBlock = 256;
    int blocksPerGrid = N/256;
    add_CPU<<<blocksPerGrid,threadsPerBlock>>>(a, b, c);
    hipDeviceSynchronize();

    // ===== Copy data from device memory to host memory
    err = hipMemcpy(v3, c, sizeof(int)*N, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        perror("Error transfering vector C from device memory.\n");
        return -1;
    }
    
    // ===== Run validation
    printf("-> Validating results (should be double the index)...\n");
    for(int i = 0; i < 5; i++)
    {
        printf("host_buffer[%i]=%i\n", i, v3[i]);
    }

    // ===== Free device buffers
    free(v3);
    hipFree(a);
    hipFree(b);
    hipFree(c);

    // ===== Done
    printf("-> Done :)\n");

    return 0;
}