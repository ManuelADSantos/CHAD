// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 18/10/2023
// ============================================================================
// -> Compile
// nvcc -o ex3 ex3.cu
// -> Run
// ./ex3


#include <hip/hip_runtime.h>
#include <stdio.h>

// ===== Device code =====
__global__ void matrix_add(int *a, int *b, int* c, int N)
{
    // ===== Kernel Code
    // Get row and column of matrix
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Sum matrices a and b and store result in matrix c
    c[row * N + col] = a[row * N + col] + b[row * N + col];
}

// ===== Host code =====
int main()
{
    int N = 16, err = 0, N_show = 16;
    int M1[N][N], M2[N][N], M3[N][N];
    int *a, *b, *c = NULL;

    // ===== Initialize host matrices
    for(int i = 0; i < N; i++)
    {
        for (int j = 0; j < N; j++)
        {   
            M1[i][j] = i + j;
            M2[i][j] = i + j;
        }
    }

    // ===== Print host matrices
    printf("\nMatrix M1:\n");
    for(int i = 0; i < N_show; i++)
    {
        for (int j = 0; j < N_show; j++)
        {
            printf("%2d ", M1[i][j]);
        }
        printf("\n");
    }
    printf("\nMatrix M2:\n");
    for(int i = 0; i < N_show; i++)
    {
        for (int j = 0; j < N_show; j++)
        {
            printf("%2d ", M2[i][j]);
        }
        printf("\n");
    }

    // ==== Allocate matrices to sum and result in device memory
    err = hipMalloc(&a, sizeof(int)*N*N);
    err = hipMalloc(&b, sizeof(int)*N*N);
    err = hipMalloc(&c, sizeof(int)*N*N);
    if(err != hipSuccess)
    {
        perror("Memory allocation for vector A, B or C failed in device\n");
        return -1;
    }

    // ===== Copy data from host memory to device memory
    err = hipMemcpy(a, M1, sizeof(int)*N*N, hipMemcpyHostToDevice);
    err = hipMemcpy(b, M2, sizeof(int)*N*N, hipMemcpyHostToDevice);
    if(err != hipSuccess)
    {
        perror("Error transfering vector M1 or M2 from host memory to device.\n");
        return -1;
    }

    // ===== Launch device function
    dim3 blocksPerGrid(1, 1);
    dim3 threadsPerBlock(N, N);
    matrix_add<<<blocksPerGrid,threadsPerBlock>>>(a, b, c, N);
    hipDeviceSynchronize();

    // ===== Copy result from device memory to host memory
    err = hipMemcpy(M3, c, sizeof(int)*N*N, hipMemcpyDeviceToHost);
    if(err != hipSuccess)
    {
        perror("Error transfering vector C from device memory.\n");
        return -1;
    }

    // ===== Run validation
    printf("\nMatrix M3:\n");
    for(int i = 0; i < N_show; i++)
    {
        for (int j = 0; j < N_show; j++)
        {
            printf("%2d ", M3[i][j]);
        }
        printf("\n");
    }

    // ===== Free device buffers   
    hipFree(a);
    hipFree(b);
    hipFree(c);

    // ===== Done
    printf("\n-> Done :)\n");

    return 0;
}