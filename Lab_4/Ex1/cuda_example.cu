// ============================================================================
// Programmer: Manuel Santos 2019231352
// Date: 17/10/2023
// ============================================================================
// -> Compile
// nvcc -o cuda_example cuda_example.cu
// -> Run
// ./cuda_example


#include <hip/hip_runtime.h>
#include <stdio.h>
// Device code
__global__ void device_func_name(int * device_buffer, int N)
{
    // Thread identifier (1 dimensional problem)
    int index= threadIdx.x + blockIdx.x * blockDim.x;
    // CODE
    if(index<N)
        device_buffer[index]=index*2;
}

int main()
{
    int N=1024;
    int err=0;
    // Allocate buffer in the device
    int *device_buffer=NULL;
    err=hipMalloc(&device_buffer, sizeof(int)*N);
    if(err!=hipSuccess)
    {
        printf("Error allocating device memory.\n");
    }
    // Launch device function
    int threadsPerBlock=256;
    int blocksPerGrid=N/256;
    device_func_name<<<blocksPerGrid,threadsPerBlock>>>(device_buffer,N);
    // Copy data from device memory to host memory
    int * host_buffer=(int *)malloc(sizeof(int)*N);
    err=hipMemcpy(host_buffer,device_buffer, sizeof(int)*N,hipMemcpyDeviceToHost);
    if(err!=hipSuccess)
    {
        printf("Error transfering data from device memory.\n");
    }
    printf("host_buffer[%i]=%i\n", N/2+1, host_buffer[N/2+1]);
    // Free device buffers
    hipFree(device_buffer);

    free(host_buffer);
    return 0;
}
